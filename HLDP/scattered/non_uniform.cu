#include "hip/hip_runtime.h"

#include "headers/myHeaders.h"
#include "headers/myUtilityFunctions.h"

using namespace std;


int main(int argc, char const *argv[])
{  
   
   //create array at host : initialize accordingly
   cellType *h_array;
   h_array = create_array_host();

   //initialize base row arguments (cellType *h_array, int rowNumber, int mode, int value)
   // : mode =1 for random initialization, put any value in that case
   initialize_this_row(h_array, 0, 0, 0);
   initialize_this_col(h_array, 0, 0, 0);
    
   //Create array at device
   cellType *d_array;
   hipMalloc((void**) &d_array, sizeof(cellType)*(nRows*TOTAL_COLS));

   //copy host array to device arrray, if needed
   copy_host_to_device(h_array, d_array);

   // create/initialize and transfer other resources and pass to the function
   //int W = nRows;

   /*int h_v[5] = {0, 10, 40, 30, 50};
   int h_w[5] = {0, 5, 4, 6, 3};*/
   
   int *h_v = create_array_host_1D(nCols);
   initialize_this_1D_array(h_v, nCols);
   int *d_v;
   hipMalloc((void**) &d_v, sizeof(int)*(nCols));
   copy_host_to_device_1D(h_v, d_v, nCols);

   int *h_w = create_array_host_1D(nRows);
   initialize_this_1D_array(h_w, nRows);
   int *d_w;
   hipMalloc((void**) &d_w, sizeof(int)*(nRows));
   copy_host_to_device_1D(h_w, d_w, nRows);
   
   //configure kernel
   configure_kernal(TOTAL_COLS);

   GpuTimer phase1;
   phase1.Start();
   
   //execute on GPU, row by row
   for (int i = 1; i < nRows; ++i)
   {
      update_array_gpu_hybrid<<<dim3(g,1,1), dim3(x,1,1)>>>(i, nCols, d_array, d_v, d_w);
   }

   phase1.Stop();
   cout <<"Time (Basic GPU): " <<phase1.Elapsed()<< " Milli Seconds\n";

   //copy back to cpu
    copy_device_to_host(h_array, d_array);
   
   //Access the resultant matrix : dump into output file
   //write_array_console(h_array);
   ofstream myfile ("files_output/o_gpu_basic.txt");
   //write_array_file(h_array, myfile);
   
   
   return 0;
}

__global__ void update_array_gpu_hybrid(int i, int numberOfThreadsRequired, cellType *d_array, int *d_v, int *d_w)
{
   long j=blockIdx.x *blockDim.x + threadIdx.x + 1;
   __shared__ cellType sharedArray [nCols];

   if (j>= numberOfThreadsRequired || j < 1)
      {}
   else
   {  
      int j_ext_1 = j - d_v[j+1];
      int j_ext_2 = j + d_v[j-2];
      int j_ext_3 = j - d_v[j+3];
      int j_ext_4 = j + d_v[j+7];
      int j_ext_5 = j - d_v[j+3];
      int j_ext_6 = j + d_v[j+7];

      if (j_ext_1 < 0 || j_ext_1 > nCols-1)
         j_ext_1 = 0;
      if (j_ext_2 < 0 || j_ext_2 > nCols-1)
         j_ext_2 = 0;
      if (j_ext_3 < 0 || j_ext_3 > nCols-1)
         j_ext_3 = 0;
      if (j_ext_4 < 0 || j_ext_4 > nCols-1)
         j_ext_4 = 0;
      if (j_ext_5 < 0 || j_ext_5 > nCols-1)
         j_ext_5 = 0;
      if (j_ext_6 < 0 || j_ext_6 > nCols-1)
         j_ext_6 = 0;

      sharedArray[j] = d_array(i-1,j);
      if (j % 5 == 0)
      {
         
      sharedArray[j] = sharedArray[j_ext_1] + sharedArray[j_ext_2] + sharedArray[j_ext_3] + sharedArray[j_ext_4] + sharedArray[j_ext_5] + sharedArray[j_ext_6];
      
      //d_array(i,j) = d_array(i-1,j_ext_1) + d_array(i-1,j_ext_2) + d_array(i-1,j_ext_3) + d_array(i-1,j_ext_4) + d_array(i-1,j_ext_5) + d_array(i-1,j_ext_6) ;
      }
      
      
   }

   
}
