#include "hip/hip_runtime.h"

#include "headers/myHeaders.h"
#include "headers/myUtilityFunctions.h"

using namespace std;


int main(int argc, char const *argv[])
{	
	
	//create array at host : initialize accordingly
	cellType *h_array;
	h_array = create_array_host();

	//initialize base row arguments (cellType *h_array, int rowNumber, int mode, int value)
	// : mode =1 for random initialization, put any value in that case
	initialize_this_row(h_array, 0, 0, 0);
	initialize_this_col(h_array, 0, 0, 0);
	 
	//Create array at device
	cellType *d_array;
	hipMalloc((void**) &d_array, sizeof(cellType)*(nRows*TOTAL_COLS));

	//copy host array to device arrray, if needed
	copy_host_to_device(h_array, d_array);

	// create/initialize and transfer other resources and pass to the function
	//int W = nRows;

	/*int h_v[5] = {0, 10, 40, 30, 50};
	int h_w[5] = {0, 5, 4, 6, 3};*/
	
	int *h_v = create_array_host_1D(nRows);
	initialize_this_1D_array(h_v, nRows);
	int *d_v;
	hipMalloc((void**) &d_v, sizeof(int)*(nRows));
	copy_host_to_device_1D(h_v, d_v, nRows);

	int *h_w = create_array_host_1D(nRows);
	initialize_this_1D_array(h_w, nRows);
	int *d_w;
	hipMalloc((void**) &d_w, sizeof(int)*(nRows));
	copy_host_to_device_1D(h_w, d_w, nRows);
	
	//configure kernel
	configure_kernal(TOTAL_COLS);

	GpuTimer phase1;
	phase1.Start();
	
	//execute on GPU, row by row
   for (int i = 1; i < nRows; ++i)
   {
	   	update_array_gpu<<<dim3(g,1,1), dim3(x,1,1)>>>(i, nCols, d_array, d_v, d_w);
   }

	phase1.Stop();
	cout <<"Time (Basic GPU): " <<phase1.Elapsed()<< " Milli Seconds\n";

	//copy back to cpu
    copy_device_to_host(h_array, d_array);
	
	//Access the resultant matrix : dump into output file
	//write_array_console(h_array);
	//ofstream myfile ("files_output/o_gpu_basic.txt");
	//write_array_file(h_array, myfile);
	
	
	return 0;
}

__global__ void update_array_gpu(int i, int numberOfThreadsRequired, cellType *d_array, int *d_v, int *d_w)
{
   long j=blockIdx.x *blockDim.x + threadIdx.x + 1;

   if (j>= numberOfThreadsRequired || j < 1)
      {}
   else
   {
		int j_ext = j - d_w[i];
		if (j_ext <= 0)
			j_ext = 0;
		int a = d_array(i-1,j);
		int b = d_v[i] + d_array(i-1,j_ext);
		(( (d_w[i]) > j || (a >= b)) ? d_array(i,j) = a : d_array(i,j) = b );
	}
}
