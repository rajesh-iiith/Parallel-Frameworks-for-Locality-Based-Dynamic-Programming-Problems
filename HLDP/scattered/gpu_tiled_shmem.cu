#include "hip/hip_runtime.h"

#include "headers/myHeaders.h"
#include "headers/myUtilityFunctions.h"

using namespace std;

__global__ void update_array_gpu_tiled(int mode, int i, cellType *d_array, int *d_v, int *d_w, int dependencyWidthLeft)
{
   int myBlockId = (blockIdx.x) + 1;
   int r,c;

   //generate hanging points (r,c)
   if (mode == 1)
   {
    	r = 1 + (i-1) * TILE_ROWS - ( (myBlockId - 1) * TILE_ROWS);
    	c = 1 + (myBlockId - 1) * TILE_COLS;
    }
   else
   {
   		int midPointIteration = nCols / TILE_COLS;
   		r = (1 + (midPointIteration - 1) * TILE_ROWS) - ((myBlockId - 1) * TILE_ROWS);
		c = 1 + ((midPointIteration - i) * (TILE_COLS)) + ((myBlockId - 1)* TILE_COLS);
   }

	//correct till r,c
   //generate my location and process the block
   // myCol is the column assigned to thread x of a given block
   int myCol = c + threadIdx.x - dependencyWidthLeft;
  __shared__ cellType sharedArray [TILE_COLS + 10];
   
   for (int iter = 1; iter <= TILE_ROWS; ++iter)
   {
   	int myRow = r + (iter-1);
   	//copy
   	if (myCol >= 0)
   		sharedArray[threadIdx.x] = d_array(myRow-1, myCol);
   	else
   		sharedArray[threadIdx.x] = d_array(myRow-1, 0);
   	__syncthreads();
   	
   	if (threadIdx.x >= dependencyWidthLeft)
   	{
   		int a = sharedArray[threadIdx.x];
		int b = d_v[myRow] + sharedArray[threadIdx.x - d_w[myRow]];
		(( (d_w[myRow] > myCol) || (a >= b)) ? d_array(myRow,myCol) = a : d_array(myRow,myCol) = b );
   	  //d_array(myRow, myCol) = (sharedArray[threadIdx.x] + sharedArray[threadIdx.x -1 ] + sharedArray[threadIdx.x -2] + 1) % 10;
   	}
   	__syncthreads();
   }
}



int main(int argc, char const *argv[])
{	
	
	//create array at host : initialize accordingly
	cellType *h_array;
	h_array = create_array_host();

	//initialize base row arguments (cellType *h_array, int rowNumber, int mode, int value)
	// : mode =1 for random initialization, put any value in that case
	initialize_this_row(h_array, 0, 0, 0);
	initialize_this_col(h_array, 0, 0, 0);
	 
	//Create array at device
	cellType *d_array;
	hipMalloc((void**) &d_array, sizeof(cellType)*(nRows*TOTAL_COLS));

	//copy host array to device arrray, if needed
	copy_host_to_device(h_array, d_array);

	// create/initialize and transfer other resources and pass to the function
	//int W = nRows;

	/*int h_v[5] = {0, 10, 40, 30, 50};
	int h_w[5] = {0, 5, 4, 6, 3};*/
	
	int *h_v = create_array_host_1D(nRows);
	initialize_this_1D_array(h_v, nRows);
	int *d_v;
	hipMalloc((void**) &d_v, sizeof(int)*(nRows));
	copy_host_to_device_1D(h_v, d_v, nRows);

	int *h_w = create_array_host_1D(nRows);
	initialize_this_1D_array(h_w, nRows);
	int *d_w;
	hipMalloc((void**) &d_w, sizeof(int)*(nRows));
	copy_host_to_device_1D(h_w, d_w, nRows);

	GpuTimer phase1;
	phase1.Start();
	
	int dependencyWidthLeft = 10;
	//create a wrapper to design tiling iterations
	int ThreadsPerBlock = dependencyWidthLeft + TILE_COLS;
	for (int i = 1; i <= (nRows/TILE_ROWS); i++)
	{
		//number of blocks in the ith iteration will be equal to i
		update_array_gpu_tiled<<<dim3(i,1,1), dim3(ThreadsPerBlock,1,1)>>>(1, i, d_array, d_v, d_w, dependencyWidthLeft);
	}
	for (int i = (nRows/TILE_ROWS)-1; i >= 1; i--)
	{
		//number of blocks in the ith iteration will be equal to i
		update_array_gpu_tiled<<<dim3(i,1,1), dim3(ThreadsPerBlock,1,1)>>>(2, i, d_array, d_v, d_w, dependencyWidthLeft);
	}
	
	phase1.Stop();
	cout <<"Time (Tiled GPU): " <<phase1.Elapsed()<< " Milli Seconds\n";

	//copy back to cpu
    copy_device_to_host(h_array, d_array);
	
	//Access the resultant matrix : dump into output file
	//write_array_console(h_array);
	//ofstream myfile ("files_output/o_gpu_tiled_shmem.txt");
	//write_array_file(h_array, myfile);
	
	
	return 0;
}

