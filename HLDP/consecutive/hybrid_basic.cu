
#include <hip/hip_runtime.h>
/*
#include "headers/myHeaders.h"
#include "headers/myUtilityFunctions.h"

using namespace std;


int main(int argc, char const *argv[])
{	
	
	//create array at host : initialize accordingly
	cellType *h_array;
	h_array = create_array_host();

	//initialize base row arguments (cellType *h_array, int rowNumber, int mode, int value)
	// : mode =1 for random initialization, put any value in that case
	initialize_this_row(h_array, 0, 1, -1);
	 
	//Create array at device
	cellType *d_array;
	cudaMalloc((void**) &d_array, sizeof(cellType)*(nRows*TOTAL_COLS));

	//copy host array to device arrray, if needed
	copy_host_to_device(h_array, d_array);
	
	//configure kernel
	configure_kernal(TOTAL_COLS);

	GpuTimer phase1;
	phase1.Start();
	
	//execute on GPU, row by row
   for (int i = 1; i < nRows; ++i)
   {
	   	update_array_gpu_hybrid<<<dim3(g,1,1), dim3(x,1,1)>>>(i, TOTAL_COLS, d_array);
	   	//update_array_cpu(i, h_array);
   }

	phase1.Stop();
	cout <<"Time (Basic GPU): " <<phase1.Elapsed()<< " Milli Seconds\n";

	//copy back to cpu
    copy_device_to_host(h_array, d_array);
	
	//Access the resultant matrix : dump into output file
	//write_array_console(h_array);
	ofstream myfile ("files_output/o_gpu_basic.txt");
	write_array_file(h_array, myfile);
	
	
	return 0;
}

__global__ void update_array_gpu_hybrid(int i, int numberOfThreadsRequired, cellType *d_array )
{
   long j=blockIdx.x *blockDim.x + threadIdx.x + 1;
   
   if (j>= numberOfThreadsRequired || j < dependencyWidthLeft)
      {}
   else
   {
   	d_array(i,j)= (d_array(i-1,j) + d_array(i-1,j-1) + d_array(i-1,j-2) + 1) % 10;
   }
}

void update_array_cpu_hybrid(int i, cellType *h_array)
{
		//#pragma omp parallel for
		for (int j = dependencyWidthLeft; j < nCols + dependencyWidthLeft ; ++j)
	   	{	
	   		h_array(i,j)= (h_array(i-1,j) + h_array(i-1,j-1) +  h_array(i-1,j-2) + 1) % 10;
		}
}*/