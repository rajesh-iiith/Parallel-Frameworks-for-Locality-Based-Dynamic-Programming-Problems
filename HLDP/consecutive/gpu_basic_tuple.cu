#include "hip/hip_runtime.h"

#include "headers/myHeaders.h"
#include "headers/myUtilityFunctions.h"

using namespace std;


int main(int argc, char const *argv[])
{	
	
	//create array at host : initialize accordingly
	cellType *h_array;
	h_array = create_array_host();

 
	//Create array at device
	cellType *d_array;
	hipMalloc((void**) &d_array, sizeof(cellType)*(nRows*TOTAL_COLS));

	//copy host array to device arrray, if needed
	copy_host_to_device(h_array, d_array);
	
	//configure kernel
	configure_kernal(TOTAL_COLS);

	GpuTimer phase1;
	phase1.Start();
	
	//execute on GPU, row by row
   for (int i = 1; i < nRows; ++i)
   {
	   	update_array_gpu<<<dim3(g,1,1), dim3(x,1,1)>>>(i, TOTAL_COLS, d_array);
   }

	phase1.Stop();
	cout <<"Time (basic GPU): " <<phase1.Elapsed()<< " Milli Seconds\n";

	//copy back to cpu
    copy_device_to_host(h_array, d_array);
	
	//Access the resultant matrix : dump into output file
	//write_array_console(h_array);
	//write_array_file(h_array, "../files_output/output_s.txt");
	
	
	return 0;
}

__global__ void update_array_gpu(int i, int numberOfThreadsRequired, cellType *d_array )
{
   long j=blockIdx.x *blockDim.x + threadIdx.x + 1;
   
   if (j>= numberOfThreadsRequired || j < dependencyWidthLeft)
      {}
   else
   {
   	d_array(i,j).value1= (d_array(i-1,j).value1 + d_array(i-1,j-1).value1) * d_array(i-1,j-2).value1 / (d_array(i-1,j-3).value1 + d_array(i-1,j-4).value1 +d_array(i-1,j-5).value1 +d_array(i-1,j-6).value1 +1);
   	d_array(i,j).value2= (d_array(i-1,j).value2 + d_array(i-1,j-1).value2) * d_array(i-1,j-2).value2 / (d_array(i-1,j-3).value2 + d_array(i-1,j-4).value2 +d_array(i-1,j-5).value2 +d_array(i-1,j-6).value2 +1);

   	//d_array(i,j)= (d_array(i-1,j) + d_array(i-1,j-1)) * d_array(i-1,j-2) / (d_array(i-1,j-3) + d_array(i-1,j-4) +d_array(i-1,j-5) +d_array(i-1,j-6) +1);

   }
}
