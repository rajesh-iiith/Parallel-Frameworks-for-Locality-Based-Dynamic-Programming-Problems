#include "hip/hip_runtime.h"

#include "headers/myHeaders.h"
#include "headers/myUtilityFunctions.h"

using namespace std;


int main(int argc, char const *argv[])
{	
	
	//create array at host : initialize accordingly
	cellType *h_array;
	h_array = create_array_host();

	//initialize base row arguments (cellType *h_array, int rowNumber, int mode, int value)
	// : mode =1 for random initialization, put any value in that case
	initialize_this_row(h_array, 0, 1, -1);
	 
	//Create array at device
	cellType *d_array;
	hipMalloc((void**) &d_array, sizeof(cellType)*(nRows*TOTAL_COLS));

	//copy host array to device arrray, if needed
	copy_host_to_device(h_array, d_array);
	
	//configure kernel
	configure_kernal(TOTAL_COLS);

	GpuTimer phase1;
	phase1.Start();
	
	//execute on GPU, row by row
   for (int i = 1; i < nRows; ++i)
   {
	   	update_array_gpu<<<dim3(g,1,1), dim3(x,1,1)>>>(i, TOTAL_COLS, d_array);
   }

	phase1.Stop();
	cout <<"Time (Basic GPU): " <<phase1.Elapsed()<< " Milli Seconds\n";

	//copy back to cpu
    copy_device_to_host(h_array, d_array);
	
	//Access the resultant matrix : dump into output file
	//write_array_console(h_array);
	//ofstream myfile ("files_output/o_gpu_basic.txt");
	//write_array_file(h_array, myfile);
	
	
	return 0;
}

__global__ void update_array_gpu(int i, int numberOfThreadsRequired, cellType *d_array )
{
   long j=blockIdx.x *blockDim.x + threadIdx.x + 1;
   
   if (j>= numberOfThreadsRequired || j < dependencyWidthLeft)
      {}
   else
   {
   			int a = (d_array(i,j).value1 + d_array(i-1,j).value1) / (d_array(i-1,j).value2 + 3) ;
			int b = (d_array(i,j).value1 + d_array(i-1,j-1).value1) / (d_array(i-1,j-1).value2 + 3);
			int c = (d_array(i,j).value1 + d_array(i-1,j-2).value1) + (d_array(i-1,j-2).value2 + 3);

			if ((a >= b) && (a >=c))
			{
				d_array(i,j).value1 = a;
				d_array(i,j).value2 = d_array(i-1,j).value2 + 3;
			}
			else
			{
				if ((b >= a) && (b >=c))
				{
					d_array(i,j).value1 = b;
					d_array(i,j).value2 = d_array(i-1,j-1).value2 + 3;
				}
				else
				{
					d_array(i,j).value1 = c;
					d_array(i,j).value2 = d_array(i-1,j-2).value2 + 3;
				}
			}
   }
}
