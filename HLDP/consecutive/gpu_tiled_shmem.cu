#include "hip/hip_runtime.h"

#include "headers/myHeaders.h"
#include "headers/myUtilityFunctions.h"

using namespace std;

__global__ void update_array_gpu_tiled(int mode, int i, cellType *d_array)
{
   int myBlockId = (blockIdx.x) + 1;
   int r,c;

   //generate hanging points (r,c)
   if (mode == 1)
   {
    	r = 1 + (i-1) * TILE_ROWS - ( (myBlockId - 1) * TILE_ROWS);
    	c = (myBlockId - 1) * TILE_COLS + dependencyWidthLeft;
    }
   else
   {
   		int midPointIteration = nCols / TILE_COLS;
   		r = (1 + (midPointIteration - 1) * TILE_ROWS) - ((myBlockId - 1) * TILE_ROWS);
		c = ((midPointIteration - i) * (TILE_COLS)) + ((myBlockId - 1)* TILE_COLS) + dependencyWidthLeft ;
   }

   //generate my location and process the block
   // myCol is the column assigned to thread x of a given block
   int myCol = c + threadIdx.x - dependencyWidthLeft;
   
   __shared__ cellType sharedArray [TILE_COLS + dependencyWidthLeft];
   

   for (int iter = 1; iter <= TILE_ROWS; ++iter)
   {
   	int myRow = r + (iter-1);
   	sharedArray[threadIdx.x] = d_array(myRow-1, myCol);
   	__syncthreads();
   	if (threadIdx.x >= dependencyWidthLeft)
   	{
   			int a = (d_array(myRow, myCol).value1 + sharedArray[threadIdx.x].value1) / (sharedArray[threadIdx.x].value2 + 3) ;
			int b = (d_array(myRow, myCol).value1 + sharedArray[threadIdx.x - 1].value1) / (sharedArray[threadIdx.x - 1].value2 + 3);
			int c = (d_array(myRow, myCol).value1 + sharedArray[threadIdx.x - 2].value1) + (sharedArray[threadIdx.x - 2].value2 + 3);

			if ((a >= b) && (a >=c))
			{
				d_array(myRow, myCol).value1 = a;
				d_array(myRow, myCol).value2 = sharedArray[threadIdx.x].value2 + 3;
			}
			else
			{
				if ((b >= a) && (b >=c))
				{
					d_array(myRow, myCol).value1 = b;
					d_array(myRow, myCol).value2 = sharedArray[threadIdx.x - 1].value2 + 3;
				}
				else
				{
					d_array(myRow, myCol).value1 = c;
					d_array(myRow, myCol).value2 = sharedArray[threadIdx.x - 2].value2 + 3;
				}
			}
   	}
   	__syncthreads();
   }
}



int main(int argc, char const *argv[])
{	
	
	//create array at host : initialize accordingly
	cellType *h_array;
	h_array = create_array_host();

	//initialize base row arguments (cellType *h_array, int rowNumber, int mode, int value)
	// : mode =1 for random initialization, put any value in that case
	initialize_this_row(h_array, 0, 1, -1);

	 
	//Create array at device
	cellType *d_array;
	hipMalloc((void**) &d_array, sizeof(cellType)*((nRows) * TOTAL_COLS));

	//copy host array to device arrray, if needed
	copy_host_to_device(h_array, d_array);

	GpuTimer phase1;
	phase1.Start();
	
	//create a wrapper to design tiling iterations
	int ThreadsPerBlock = dependencyWidthLeft + TILE_COLS;
	for (int i = 1; i <= (nRows/TILE_ROWS); i++)
	{
		//number of blocks in the ith iteration will be equal to i
		update_array_gpu_tiled<<<dim3(i,1,1), dim3(ThreadsPerBlock,1,1)>>>(1, i, d_array);
	}
	for (int i = (nRows/TILE_ROWS)-1; i >= 1; i--)
	{
		//number of blocks in the ith iteration will be equal to i
		update_array_gpu_tiled<<<dim3(i,1,1), dim3(ThreadsPerBlock,1,1)>>>(2, i, d_array);
	}
	
	phase1.Stop();
	cout <<"Time (Tiled GPU): " <<phase1.Elapsed()<< " Milli Seconds\n";

	//copy back to cpu
    copy_device_to_host(h_array, d_array);
	
	//Access the resultant matrix : dump into output file
	//write_array_console(h_array);
	//ofstream myfile ("files_output/o_gpu_tiled_shmem.txt");
	//write_array_file(h_array, myfile);
	
	
	return 0;
}

