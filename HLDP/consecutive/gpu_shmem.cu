#include "hip/hip_runtime.h"

#include "headers/myHeaders.h"
#include "headers/myUtilityFunctions.h"

using namespace std;


int main(int argc, char const *argv[])
{	
	
	//create array at host : initialize accordingly
	cellType *h_array;
	h_array = create_array_host();
	 
	//Create array at device
	cellType *d_array;
	hipMalloc((void**) &d_array, sizeof(cellType)*(nRows*TOTAL_COLS));

	//copy host array to device arrray, if needed
	copy_host_to_device(h_array, d_array);
	
	//configure kernel
	//cwThreads do both copy and work, cThreads just do copy 
	int cwThreadsPerBlock = BLOCK_SIZE - dependencyWidthLeft;
	int threadsToBeLaunched = (((nCols / cwThreadsPerBlock) + 1) * BLOCK_SIZE);
	int lastGloballyActiveThread = TOTAL_COLS + ((nCols / cwThreadsPerBlock) * dependencyWidthLeft);
	configure_kernal_shmem(threadsToBeLaunched);
	cout << dependencyWidthLeft <<","<< threadsToBeLaunched << "," << g << "," << x << "\n"; 

	GpuTimer phase1;
	phase1.Start();
	
	//execute on GPU, row by row
   for (int i = 1; i < nRows; ++i)
   {
	   	update_array_gpu_shmem<<<dim3(g,1,1), dim3(x,1,1)>>>(i, lastGloballyActiveThread, cwThreadsPerBlock, d_array);
   }

	phase1.Stop();
	cout <<"Time (shared mamory GPU): " <<phase1.Elapsed()<< " Milli Seconds\n";

	//copy back to cpu
    copy_device_to_host(h_array, d_array);
	
	//Access the resultant matrix : dump into output file
	//write_array_console(h_array);
	//write_array_file(h_array, "../files_output/output_s.txt");
	
	
	return 0;
}

__global__ void update_array_gpu_shmem(int i, int lastGloballyActiveThread, int cwThreadsPerBlock, cellType *d_array )
{
   //create shared array
   __shared__ cellType d_array_shared[BLOCK_SIZE];
   
   //copy appropriate chunk of global array into shared array 
   long local_index = threadIdx.x; //logically sud be +1
   long j = local_index + (cwThreadsPerBlock*blockIdx.x);
   long global_index = blockIdx.x *blockDim.x + threadIdx.x + 1;
   
   d_array_shared[local_index] = d_array(i-1,j);

   //synch threads after copy
   //__syncthreads();

   if (global_index >= lastGloballyActiveThread + 1 || threadIdx.x <= (dependencyWidthLeft - 1))
      {}
   else
   {
	//write back the result into global array


	d_array(i,j) =  (d_array_shared[local_index]  + d_array_shared[local_index -1]) * d_array_shared[local_index -2]  / (d_array_shared[local_index -3] + d_array_shared[local_index -4] + d_array_shared[local_index -5] + d_array_shared[local_index -6] + 1);
	
   }
}