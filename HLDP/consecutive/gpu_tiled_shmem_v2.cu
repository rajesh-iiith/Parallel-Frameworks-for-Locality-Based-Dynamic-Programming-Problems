#include "hip/hip_runtime.h"

#include "headers/myHeaders.h"
#include "headers/myUtilityFunctions.h"

using namespace std;

__global__ void update_array_gpu_tiled(int mode, int i, cellType *d_array)
{
   int myBlockId = (blockIdx.x) + 1;
   int r,c;

   //generate hanging points (r,c)
   if (mode == 1)
   {
    	r = 1 + (i-1) * TILE_ROWS - ( (myBlockId - 1) * TILE_ROWS);
    	c = (myBlockId - 1) * TILE_COLS + dependencyWidthLeft;
    }
   else
   {
   		int midPointIteration = nCols / TILE_COLS;
   		r = (1 + (midPointIteration - 1) * TILE_ROWS) - ((myBlockId - 1) * TILE_ROWS);
		c = ((midPointIteration - i) * (TILE_COLS)) + ((myBlockId - 1)* TILE_COLS) + dependencyWidthLeft ;
   }

   //generate my location and process the block
   // myCol is the column assigned to thread x of a given block
   int myCol = c + threadIdx.x - dependencyWidthLeft;

   //named even/odd by the iteration number in which array is used
   const int columnWidth = TILE_COLS + dependencyWidthLeft;
   __shared__ cellType jointArray [2 * (columnWidth)];
   
   // iter: 1
   jointArray[columnWidth + threadIdx.x] = d_array(r-1, myCol);
   __syncthreads();
   if (threadIdx.x >= dependencyWidthLeft)
   	{
		jointArray[threadIdx.x] = (jointArray[columnWidth + threadIdx.x] + jointArray[columnWidth + threadIdx.x -1] + jointArray[columnWidth + threadIdx.x -2] + 1) % 10;	
		d_array(r, myCol) = jointArray[threadIdx.x];
   	}
   	__syncthreads();

   	// iter: 2 and onwards
   for (int iter = 2; iter <= TILE_ROWS; ++iter)
   {
	   	int myRow = r + (iter-1);
	   	int oo_ee = -((iter % 2) - 1 ) * columnWidth;
	   	int oe_eo = (iter % 2) * columnWidth;
	   	//step 1: copy the required portion from global and remaining portion from shared
	   	
   		if ( (threadIdx.x < dependencyWidthLeft) )	
   		{
   			jointArray[oe_eo + threadIdx.x] = d_array(myRow-1, myCol);
   		}
	 	__syncthreads();

	   	// step 2: operate and don't forget to copy shared and global memory both.
	   	if (threadIdx.x >= dependencyWidthLeft)
	   	{	
	   		jointArray[oo_ee + threadIdx.x] = (jointArray[oe_eo + threadIdx.x] + jointArray[oe_eo + threadIdx.x -1] + jointArray[oe_eo + threadIdx.x -2] + 1) %10;
	   		d_array(myRow, myCol) = jointArray[oo_ee + threadIdx.x];		
	   	}
	   	__syncthreads();

   }
}



int main(int argc, char const *argv[])
{	
	
	//create array at host : initialize accordingly
	cellType *h_array;
	h_array = create_array_host();

	//initialize base row arguments (cellType *h_array, int rowNumber, int mode, int value)
	// : mode =1 for random initialization, put any value in that case
	initialize_this_row(h_array, 0, 1, -1);

	 
	//Create array at device
	cellType *d_array;
	hipMalloc((void**) &d_array, sizeof(cellType)*((nRows) * TOTAL_COLS));

	//copy host array to device arrray, if needed
	copy_host_to_device(h_array, d_array);

	GpuTimer phase1;
	phase1.Start();
	
	//create a wrapper to design tiling iterations
	int ThreadsPerBlock = dependencyWidthLeft + TILE_COLS;
	for (int i = 1; i <= (nRows/TILE_ROWS); i++)
	{
		//number of blocks in the ith iteration will be equal to i
		update_array_gpu_tiled<<<dim3(i,1,1), dim3(ThreadsPerBlock,1,1)>>>(1, i, d_array);
	}
	for (int i = (nRows/TILE_ROWS)-1; i >= 1; i--)
	{
		//number of blocks in the ith iteration will be equal to i
		update_array_gpu_tiled<<<dim3(i,1,1), dim3(ThreadsPerBlock,1,1)>>>(2, i, d_array);
	}
	
	phase1.Stop();
	cout <<"Time (Tiled GPU): " <<phase1.Elapsed()<< " Milli Seconds\n";

	//copy back to cpu
    copy_device_to_host(h_array, d_array);
	
	//Access the resultant matrix : dump into output file
	//write_array_console(h_array);
	ofstream myfile ("files_output/o_gpu_tiled_shmem_v2.txt");
	write_array_file(h_array, myfile);
	
	
	return 0;
}

