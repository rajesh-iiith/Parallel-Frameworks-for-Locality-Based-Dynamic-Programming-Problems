#include "hip/hip_runtime.h"

#include "headers/myHeaders.h"
#include "headers/myUtilityFunctions.h"

using namespace std;



__global__ void update_array_gpu_two_tables(int i, int numberOfThreadsRequired, cellType *d_array, cellType *d_T )
{
   long j=blockIdx.x *blockDim.x + threadIdx.x + 1;
   
   if (j>= numberOfThreadsRequired || j < dependencyWidthLeft)
      {}
   else
   {
   	d_array(i,j)= (d_array(i-1,j) + d_array(i-1,j-1)) * d_array(i-1,j-2) / (d_array(i-1,j-3) + d_array(i-1,j-4) +d_array(i-1,j-5) +d_array(i-1,j-6) +1);
   	d_T(i,j)= (d_T(i-1,j) + d_T(i-1,j-1)) * d_T(i-1,j-2) / (d_T(i-1,j-3) + d_T(i-1,j-4) +d_T(i-1,j-5) +d_T(i-1,j-6) +1);
   }
}


int main(int argc, char const *argv[])
{	
	
	//create array at host : initialize accordingly
	cellType *h_array, *h_T;
	h_array = create_array_host();
	h_T = create_array_host();

 
	//Create array at device
	cellType *d_array, *d_T;
	hipMalloc((void**) &d_array, sizeof(cellType)*(nRows*TOTAL_COLS));
	hipMalloc((void**) &d_T, sizeof(cellType)*(nRows*TOTAL_COLS));

	//copy host array to device arrray, if needed
	copy_host_to_device(h_array, d_array);
	copy_host_to_device(h_T, d_T);
	
	//configure kernel
	configure_kernal(TOTAL_COLS);

	GpuTimer phase1;
	phase1.Start();
	
	//execute on GPU, row by row
   for (int i = 1; i < nRows; ++i)
   {
	   	update_array_gpu_two_tables<<<dim3(g,1,1), dim3(x,1,1)>>>(i, TOTAL_COLS, d_array, d_T);
   }

	phase1.Stop();
	cout <<"Time (basic GPU): " <<phase1.Elapsed()<< " Milli Seconds\n";

	//copy back to cpu
    copy_device_to_host(h_array, d_array);
    copy_device_to_host(h_T, d_T);
	
	//Access the resultant matrix : dump into output file
	//write_array_console(h_array);
	//write_array_file(h_array, "../files_output/output_s.txt");
	
	
	return 0;
}
