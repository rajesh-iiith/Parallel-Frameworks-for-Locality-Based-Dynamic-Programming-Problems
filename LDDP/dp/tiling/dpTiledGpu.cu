#include "hip/hip_runtime.h"
#include "headers/myHeaders.h"

//using statements
using namespace std;

int main(int argc, char const *argv[])
{ 
  // Original array is 2d array : input image / matrix / DP table 
  arrayOrg=new int *[nRows];
  for(int z=0 ; z<nRows ; z++)
  {
    arrayOrg[z]=new int[nCols];
  }

  //flatten the array to 1d array
  h_array = (int*)calloc(nRows*nCols, sizeof(int));
  for (int i = 0; i < nRows; ++i)
  {
    for (int j = 0; j < nCols; ++j)
    {
     h_array[i*nCols + j] = arrayOrg [i][j];
    }
  }
  //free arrayOrg
  for(int z=0 ; z<nRows ; z++)
  {
    free(arrayOrg[z]);
  }
  free(arrayOrg);

  //load h_array to GPU
  // Load main resource (DP table/ Image i.e. h_array) to GPU
  int *d_array;
  hipMalloc((void**) &d_array, sizeof(int)*(nRows*nCols));
  hipMemcpy(d_array, h_array,sizeof(int)*(nRows*nCols), hipMemcpyHostToDevice);

  // Load external reseorces. e.g. subsequences in case of LCS
  //1. To CPU

   char *subsequence1 = new char[nRows];
   for (int i = 1; i < nRows; ++i)
   {
    subsequence1 [i] = rand()%4 + 67;
   }
   char *subsequence2 = new char[nCols];
   for (int i = 1; i < nRows; ++i)
   {
    subsequence2 [i] = rand()%4 + 67;
   }

   //2. To GPU

   char *d_subsequence1;
   hipMalloc((void**) &d_subsequence1, sizeof(char)*nRows);
   hipMemcpy(d_subsequence1, subsequence1,sizeof(char)*nRows, hipMemcpyHostToDevice);
   char *d_subsequence2;
   hipMalloc((void**) &d_subsequence2, sizeof(char)*nCols);
   hipMemcpy(d_subsequence2, subsequence2,sizeof(char)*nCols, hipMemcpyHostToDevice);

  // initialiation : not required if we are inputting the image : can be replaced by input code
  for (int i = 0; i < nRows; ++i)
  { 
    for (int j = 0; j < nCols; ++j)
    {
      h_array[i*nCols + j] = 0;
    }
  }


 struct timeval start, end;
 double time1 = omp_get_wtime();

 gettimeofday(&start, NULL);
 int x_start = 1;
 int y_start = 1;
 //omp_set_nested(1);
 for (int i = 1; i <= (nRows/tileLength) ; ++i)
  {
    
      operate_on_block_one_gpu <<<dim3(i,1,1), dim3(tileLength,1,1)>>> (i, d_array, x_start, y_start, d_subsequence1, d_subsequence2);
      x_start = x_start + tileLength;
  } 

    x_start = x_start - tileLength;
    y_start = y_start + tileLength;
    for (int i = (nRows/tileLength)-1; i >= 1 ; --i)
  {
    
      operate_on_block_two_gpu <<<dim3(i,1,1), dim3(tileLength,1,1)>>> (i, d_array, x_start, y_start, d_subsequence1, d_subsequence2);
      y_start = y_start + tileLength;
  } 
  gettimeofday(&end, NULL);
  double run_time = ((end.tv_sec - start.tv_sec)*1000 + (end.tv_usec - start.tv_usec)/1000.0);

  printf("Time (Blocked): %.3lf\n", run_time);
  //cout << "Time (Blocked): " << (omp_get_wtime() - time1) * 1000 << "\n";
  hipDeviceSynchronize();
  
  hipMemcpy(h_array , d_array ,sizeof(int) * (nRows*nCols), hipMemcpyDeviceToHost);

  //Access the resultant matrix or write to file
  
  /*ofstream myfile ("../output_p.txt");
  for (int i = 0; i < nRows; ++i)
  { 
    for (int j = 0; j < nCols; ++j)
    {
      myfile << h_array[i*nCols + j] << "\t";
      
    }
    myfile << "\n";
  }*/

 
  cout << "Result(Blocked on GPU): "<< h_array[nRows*nCols -1] <<"\n";
  
  return 0;
}


__global__ void operate_on_block_one_gpu (int i, int *d_array, int x_start, int y_start, char *d_subsequence1, char *d_subsequence2)
{
      long tid=blockIdx.x *blockDim.x + threadIdx.x;
      
       
          int x_my_block = x_start - (blockIdx.x) * tileLength;
          int y_my_block = y_start + (blockIdx.x) * tileLength;
          //printf("%d%d%d%d\n", i,blockIdx.x + 1, x_my_block,y_my_block);
        

        //operate_on_block (x_start, y_start, subsequence1, subsequence2);

      

                      int x_start_local = x_my_block;
                      int y_start_local = y_my_block;

                        for (int i = 1; i <= tileLength; ++i)
                        {
                          //#pragma omp parallel for
                          int j= (tid % tileLength) + 1;
                          //printf("%d-%d\n",j,i);
                          if (j <= i)
                          {
                            //printf("%d\n",j);
                            int x_my = x_start_local - (j-1)*1;
                            int y_my = y_start_local + (j-1)*1;
                            //printf("%d%d\n",x_my,y_my );
                              if (d_subsequence1 [x_my] == d_subsequence2 [y_my])
                                  {
                                    d_Z = 1 + d_B;
                                  }
                              else
                                  {
                                    ( d_A > d_C ? d_Z = d_A : d_Z = d_C );
                                  }
                          }
                            x_start_local = x_start_local + 1;
                        }

                        x_start_local = x_start_local - 1;
                        y_start_local = y_start_local + 1;

                          for (int i = tileLength-1; i >= 1 ; --i)
                          {
                                int j= (tid % tileLength) + 1;
                                if (j <= i)
                                {
                                  int x_my = x_start_local - (j-1)*1;
                                  int y_my = y_start_local + (j-1)*1;
                                  //printf("%d%d\n",x_my,y_my );
                                  if (d_subsequence1 [x_my] == d_subsequence2 [y_my])
                                    {
                                      d_Z = 1 + d_B;
                                    }
                                  else
                                    {
                                      ( d_A > d_C ? d_Z = d_A : d_Z = d_C );
                                    }

                                }

                              y_start_local = y_start_local + 1;
                          } 
    
}



__global__ void operate_on_block_two_gpu (int i, int *d_array, int x_start, int y_start, char *d_subsequence1, char *d_subsequence2)
{
      long tid=blockIdx.x *blockDim.x + threadIdx.x;
      
        int x_my_block = x_start - (blockIdx.x)*tileLength;
        int y_my_block = y_start + (blockIdx.x)*tileLength;

        //operate_on_block (x_my, y_my, subsequence1, subsequence2);

      

                      int x_start_local = x_my_block;
                      int y_start_local = y_my_block;
                        for (int i = 1; i <= tileLength; ++i)
                        {
                          int j= (tid % tileLength) + 1;
                          if (j <= i)
                          {
                            int x_my = x_start_local - (j-1)*1;
                            int y_my = y_start_local + (j-1)*1;
                              if (d_subsequence1 [x_my] == d_subsequence2 [y_my])
                                  {
                                    d_Z = 1 + d_B;
                                  }
                              else
                                  {
                                    ( d_A > d_C ? d_Z = d_A : d_Z = d_C );
                                  }
                          }
                            x_start_local = x_start_local + 1;
                        }

                        x_start_local = x_start_local - 1;
                        y_start_local = y_start_local + 1;

                          for (int i = tileLength-1; i >= 1 ; --i)
                          {
                                int j= (tid % tileLength) + 1;
                                if (j <= i)
                                {
                                  int x_my = x_start_local - (j-1)*1;
                                  int y_my = y_start_local + (j-1)*1;

                                  if (d_subsequence1 [x_my] == d_subsequence2 [y_my])
                                    {
                                      d_Z = 1 + d_B;
                                    }
                                  else
                                    {
                                      ( d_A > d_C ? d_Z = d_A : d_Z = d_C );
                                    }

                                }

                              y_start_local = y_start_local + 1;
                          } 
    
}
















