//include statements

#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include "omp.h"
#include <fstream>

//define statements
#define nRows 16385
#define nCols 16385
#define tileLength 32
// 128 is best block size in case of lcs (on the basis of experiments on 10k * 10k)
#define BLOCK_SIZE 128
#define CUTOFF_HANDOVER 500
#define CUTOFF_HYBRID 0

#define A h_array[ (x_my)*nCols + (y_my - 1) ]
#define B h_array[ (x_my - 1)*nCols + (y_my - 1) ]
#define C h_array[ (x_my - 1)*nCols + (y_my) ]
#define Z h_array[ (x_my)*nCols + (y_my) ]

#define d_A d_array[ (x_my)*nCols + (y_my - 1) ]
#define d_B d_array[ (x_my - 1)*nCols + (y_my - 1) ]
#define d_C d_array[ (x_my - 1)*nCols + (y_my) ]
#define d_Z d_array[ (x_my)*nCols + (y_my) ]



//using statements
using namespace std;

//global variables
int x,g;
int **arrayOrg;
int *h_array;
//CPU declarations
void operate_on_block_one (int i, int x_start, int y_start, char *subsequence1, char *subsequence2);
void operate_on_block_two (int i, int x_start, int y_start, char *subsequence1, char *subsequence2);
__global__ void operate_on_block_one_gpu (int i, int *d_array, int x_start, int y_start, char *d_subsequence1, char *d_subsequence2);
__global__ void operate_on_block_two_gpu (int i, int *d_array, int x_start, int y_start, char *d_subsequence1, char *d_subsequence2);

int main(int argc, char const *argv[])
{ 
  // Original array is 2d array : input image / matrix / DP table 
  arrayOrg=new int *[nRows];
  for(int z=0 ; z<nRows ; z++)
  {
    arrayOrg[z]=new int[nCols];
  }

  //flatten the array to 1d array
  h_array = (int*)calloc(nRows*nCols, sizeof(int));
  for (int i = 0; i < nRows; ++i)
  {
    for (int j = 0; j < nCols; ++j)
    {
     h_array[i*nCols + j] = arrayOrg [i][j];
    }
  }
  //free arrayOrg
  for(int z=0 ; z<nRows ; z++)
  {
    free(arrayOrg[z]);
  }
  free(arrayOrg);

  //load h_array to GPU
  // Load main resource (DP table/ Image i.e. h_array) to GPU
  int *d_array;
  hipMalloc((void**) &d_array, sizeof(int)*(nRows*nCols));
  hipMemcpy(d_array, h_array,sizeof(int)*(nRows*nCols), hipMemcpyHostToDevice);

  // Load external reseorces. e.g. subsequences in case of LCS
  //1. To CPU

   char *subsequence1 = new char[nRows];
   for (int i = 1; i < nRows; ++i)
   {
    subsequence1 [i] = rand()%4 + 67;
   }
   char *subsequence2 = new char[nCols];
   for (int i = 1; i < nRows; ++i)
   {
    subsequence2 [i] = rand()%4 + 67;
   }

   //2. To GPU

   char *d_subsequence1;
   hipMalloc((void**) &d_subsequence1, sizeof(char)*nRows);
   hipMemcpy(d_subsequence1, subsequence1,sizeof(char)*nRows, hipMemcpyHostToDevice);
   char *d_subsequence2;
   hipMalloc((void**) &d_subsequence2, sizeof(char)*nCols);
   hipMemcpy(d_subsequence2, subsequence2,sizeof(char)*nCols, hipMemcpyHostToDevice);

  // initialiation : not required if we are inputting the image : can be replaced by input code
  for (int i = 0; i < nRows; ++i)
  { 
    for (int j = 0; j < nCols; ++j)
    {
      h_array[i*nCols + j] = 0;
    }
  }

 double time1 = omp_get_wtime();
 int x_start = 1;
 int y_start = 1;
 //omp_set_nested(1);
 for (int i = 1; i <= (nRows/tileLength) ; ++i)
  {
    
      operate_on_block_one_gpu <<<dim3(i,1,1), dim3(tileLength,1,1)>>> (i, d_array, x_start, y_start, d_subsequence1, d_subsequence2);
      x_start = x_start + tileLength;
  } 

    x_start = x_start - tileLength;
    y_start = y_start + tileLength;
    for (int i = (nRows/tileLength)-1; i >= 1 ; --i)
  {
    
      operate_on_block_two_gpu <<<dim3(i,1,1), dim3(tileLength,1,1)>>> (i, d_array, x_start, y_start, d_subsequence1, d_subsequence2);
      //printf("i:%d\n", i);
      y_start = y_start + tileLength;
  } 

  cout << "Time (Blocked): " << omp_get_wtime() - time1 << "\n";
  hipMemcpy(h_array , d_array ,sizeof(int) * (nRows*nCols), hipMemcpyDeviceToHost);

  //Access the resultant matrix or write to file
  
  ofstream myfile ("output_p.txt");
  for (int i = 0; i < nRows; ++i)
  { 
    for (int j = 0; j < nCols; ++j)
    {
      myfile << h_array[i*nCols + j] << "\t";
      
    }
    myfile << "\n";
  }

  cout << "\n";
  
  cout << "Result(Blocked): "<< h_array[nRows*nCols -1] <<"\n";
  //ofstream myfile ("output_p.txt");
  /*for (int i = nRows*nCols -100; i < nRows*nCols; ++i)
  {
    myfile << h_array[i] << "\t";
  }
  */
  return 0;
}


__global__ void operate_on_block_one_gpu (int i, int *d_array, int x_start, int y_start, char *d_subsequence1, char *d_subsequence2)
{
      long tid=blockIdx.x *blockDim.x + threadIdx.x;
      
       
          int x_my_block = x_start - (blockIdx.x) * tileLength;
          int y_my_block = y_start + (blockIdx.x) * tileLength;
          //printf("%d%d%d%d\n", i,blockIdx.x + 1, x_my_block,y_my_block);
        

        //operate_on_block (x_start, y_start, subsequence1, subsequence2);

      

                      int x_start_local = x_my_block;
                      int y_start_local = y_my_block;

                        for (int i = 1; i <= tileLength; ++i)
                        {
                          //#pragma omp parallel for
                          int j= (tid % tileLength) + 1;
                          //printf("%d-%d\n",j,i);
                          if (j <= i)
                          {
                            //printf("%d\n",j);
                            int x_my = x_start_local - (j-1)*1;
                            int y_my = y_start_local + (j-1)*1;
                            //printf("%d%d\n",x_my,y_my );
                              if (d_subsequence1 [x_my] == d_subsequence2 [y_my])
                                  {
                                    d_Z = 1 + d_B;
                                  }
                              else
                                  {
                                    ( d_A > d_C ? d_Z = d_A : d_Z = d_C );
                                  }
                          }
                            x_start_local = x_start_local + 1;
                        }

                        x_start_local = x_start_local - 1;
                        y_start_local = y_start_local + 1;

                          for (int i = tileLength-1; i >= 1 ; --i)
                          {
                                int j= (tid % tileLength) + 1;
                                if (j <= i)
                                {
                                  int x_my = x_start_local - (j-1)*1;
                                  int y_my = y_start_local + (j-1)*1;
                                  //printf("%d%d\n",x_my,y_my );
                                  if (d_subsequence1 [x_my] == d_subsequence2 [y_my])
                                    {
                                      d_Z = 1 + d_B;
                                    }
                                  else
                                    {
                                      ( d_A > d_C ? d_Z = d_A : d_Z = d_C );
                                    }

                                }

                              y_start_local = y_start_local + 1;
                          } 
    
}



__global__ void operate_on_block_two_gpu (int i, int *d_array, int x_start, int y_start, char *d_subsequence1, char *d_subsequence2)
{
      long tid=blockIdx.x *blockDim.x + threadIdx.x;
      
        int x_my_block = x_start - (blockIdx.x)*tileLength;
        int y_my_block = y_start + (blockIdx.x)*tileLength;

        //operate_on_block (x_my, y_my, subsequence1, subsequence2);

      

                      int x_start_local = x_my_block;
                      int y_start_local = y_my_block;
                        for (int i = 1; i <= tileLength; ++i)
                        {
                          int j= (tid % tileLength) + 1;
                          if (j <= i)
                          {
                            int x_my = x_start_local - (j-1)*1;
                            int y_my = y_start_local + (j-1)*1;
                              if (d_subsequence1 [x_my] == d_subsequence2 [y_my])
                                  {
                                    d_Z = 1 + d_B;
                                  }
                              else
                                  {
                                    ( d_A > d_C ? d_Z = d_A : d_Z = d_C );
                                  }
                          }
                            x_start_local = x_start_local + 1;
                        }

                        x_start_local = x_start_local - 1;
                        y_start_local = y_start_local + 1;

                          for (int i = tileLength-1; i >= 1 ; --i)
                          {
                                int j= (tid % tileLength) + 1;
                                if (j <= i)
                                {
                                  int x_my = x_start_local - (j-1)*1;
                                  int y_my = y_start_local + (j-1)*1;

                                  if (d_subsequence1 [x_my] == d_subsequence2 [y_my])
                                    {
                                      d_Z = 1 + d_B;
                                    }
                                  else
                                    {
                                      ( d_A > d_C ? d_Z = d_A : d_Z = d_C );
                                    }

                                }

                              y_start_local = y_start_local + 1;
                          } 
    
}

























/*void operate_on_block_one (int i, int x_start, int y_start, char *subsequence1, char *subsequence2)
{
      #pragma omp parallel for
      for (int j = 1; j <= i; ++j)
      {
        int x_my_block = x_start - (j-1)*tileLength;
        int y_my_block = y_start + (j-1)*tileLength;

        //operate_on_block (x_start, y_start, subsequence1, subsequence2);

      

                      int x_start_local = x_my_block;
                      int y_start_local = y_my_block;
                        for (int i = 1; i <= tileLength; ++i)
                        {
                          //#pragma omp parallel for
                          for (int j = 1; j <= i; ++j)
                          {
                            int x_my = x_start_local - (j-1)*1;
                            int y_my = y_start_local + (j-1)*1;
                              if (subsequence1 [x_my] == subsequence2 [y_my])
                                  {
                                    Z = 1 + B;
                                  }
                              else
                                  {
                                    ( A > C ? Z = A : Z = C );
                                  }
                          }
                            x_start_local = x_start_local + 1;
                        }

                        x_start_local = x_start_local - 1;
                        y_start_local = y_start_local + 1;

                          for (int i = tileLength-1; i >= 1 ; --i)
                          {
                              //#pragma omp parallel for
                              for (int j = 1; j <= i; ++j)
                                {
                                  int x_my = x_start_local - (j-1)*1;
                                  int y_my = y_start_local + (j-1)*1;

                                  if (subsequence1 [x_my] == subsequence2 [y_my])
                                    {
                                      Z = 1 + B;
                                    }
                                  else
                                    {
                                      ( A > C ? Z = A : Z = C );
                                    }

                                }

                              y_start_local = y_start_local + 1;
                          } 

          }
    
}



void operate_on_block_two (int i, int x_start, int y_start, char *subsequence1, char *subsequence2)
{
      #pragma omp parallel for
      for (int j = 1; j <= i; ++j)
      {
        int x_my_block = x_start - (j-1)*tileLength;
        int y_my_block = y_start + (j-1)*tileLength;

        //operate_on_block (x_my, y_my, subsequence1, subsequence2);

      

                      int x_start_local = x_my_block;
                      int y_start_local = y_my_block;
                        for (int i = 1; i <= tileLength; ++i)
                        {
                          //#pragma omp parallel for
                          for (int j = 1; j <= i; ++j)
                          {
                            int x_my = x_start_local - (j-1)*1;
                            int y_my = y_start_local + (j-1)*1;
                              if (subsequence1 [x_my] == subsequence2 [y_my])
                                  {
                                    Z = 1 + B;
                                  }
                              else
                                  {
                                    ( A > C ? Z = A : Z = C );
                                  }
                          }
                            x_start_local = x_start_local + 1;
                        }

                        x_start_local = x_start_local - 1;
                        y_start_local = y_start_local + 1;

                          for (int i = tileLength-1; i >= 1 ; --i)
                          {
                              //#pragma omp parallel for
                              for (int j = 1; j <= i; ++j)
                                {
                                  int x_my = x_start_local - (j-1)*1;
                                  int y_my = y_start_local + (j-1)*1;

                                  if (subsequence1 [x_my] == subsequence2 [y_my])
                                    {
                                      Z = 1 + B;
                                    }
                                  else
                                    {
                                      ( A > C ? Z = A : Z = C );
                                    }

                                }

                              y_start_local = y_start_local + 1;
                          } 

          }
    
}
*/