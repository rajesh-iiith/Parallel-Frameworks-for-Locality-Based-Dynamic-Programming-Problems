//include statements

#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include "omp.h"
#include <fstream>

//define statements
#define nRows 16385
#define nCols 16385
// 128 is best block size in case of lcs (on the basis of experiments on 10k * 10k)
#define BLOCK_SIZE 128
#define CUTOFF_HANDOVER 500
#define CUTOFF_HYBRID 0

#define A1 h_array[digSize-i-1 + j]
#define B1 h_array[digSize-2*i + j]
#define C1 h_array[digSize-i + j]

#define d_A1 d_array[digSize-i-1 + j]
#define d_B1 d_array[digSize-2*i + j]
#define d_C1 d_array[digSize-i + j]

#define A2 h_array[digSize - (nRows - (i - 1)) + j]
#define B2 h_array[digSize - 2*(nRows - (i - 1)) + j]
#define C2 h_array[digSize - (nRows - (i - 1)) + 1 + j]
#define B2ex h_array[digSize - 2*(nRows - (i - 1)) + 1 + j]

#define d_A2 d_array[digSize - (nRows - (i - 1)) + j]
#define d_B2 d_array[digSize - 2*(nRows - (i - 1)) + j]
#define d_C2 d_array[digSize - (nRows - (i - 1)) + 1 + j]
#define d_B2ex d_array[digSize - 2*(nRows - (i - 1)) + 1 + j]

#define Z h_array[digSize + j]
#define d_Z d_array[digSize + j]



//using statements
using namespace std;

//global variables
int x,g;
//CPU declarations
void cpu_left (int i, int digSize, int *h_array, char *subsequence1, char *subsequence2);
void cpu_mid (int i, int digSize, int *h_array, char *subsequence1, char *subsequence2);
void cpu_bottom (int i, int digSize, int *h_array, char *subsequence1, char *subsequence2);

//GPU declarations
void configure_kernal(long);
__global__ void gpu_left (int i, int digSize, int *d_array, char *d_subsequence1, char *d_subsequence2);
__global__ void gpu_mid (int i, int digSize, int *d_array, char *d_subsequence1, char *d_subsequence2);
__global__ void gpu_bottom (int i, int digSize, int *d_array, char *d_subsequence1, char *d_subsequence2);


int main(int argc, char const *argv[])
{	
	
	
	// Original array is 2d array : input image / matrix / DP table
	int **arrayOrg; 
	arrayOrg=new int *[nRows];
	for(int z=0 ; z<nRows ; z++)
	{
		arrayOrg[z]=new int[nCols];
	}

	// Load external reseorces. e.g. subsequences in case of LCS
	//1. To CPU

	 char *subsequence1 = new char[nRows];
	 for (int i = 1; i < nRows; ++i)
	 {
	 	subsequence1 [i] = rand()%4 + 67;
	 }
	 char *subsequence2 = new char[nCols];
	 for (int i = 1; i < nRows; ++i)
	 {
	 	subsequence2 [i] = rand()%4 + 67;
	 }

	 //2. To GPU

	 char *d_subsequence1;
     hipMalloc((void**) &d_subsequence1, sizeof(char)*nRows);
     hipMemcpy(d_subsequence1, subsequence1,sizeof(char)*nRows, hipMemcpyHostToDevice);
     char *d_subsequence2;
     hipMalloc((void**) &d_subsequence2, sizeof(char)*nCols);
     hipMemcpy(d_subsequence2, subsequence2,sizeof(char)*nCols, hipMemcpyHostToDevice);

	// initialiation : not required if we are inputting the image : can be replaced by input code
	for (int i = 0; i < nRows; ++i)
	{	
		for (int j = 0; j < nCols; ++j)
		{
			arrayOrg[i][j] = 0;
		}
	}

	// memory coalescing : change to 1D array

	int *h_array = (int*)calloc(nRows*nCols, sizeof(int));
	
	int digSize = 0;
	for (int i = 0; i < nRows; ++i)
	{	
		digSize = digSize + i;
		for (int j = 0; j <= i; ++j)
		{
			h_array[digSize + j] = arrayOrg[i - j][j];
		}
		
	}
	
	for (int i = 1; i < nRows; ++i)
	{	
		digSize = digSize + nRows - (i - 1);
		for (int j = 0; j < (nRows - i); ++j)
		{
			h_array[digSize + j] = arrayOrg[nRows - j - 1][i+j];
		}
	}

	// Load main resource (DP table/ Image i.e. h_array) to GPU
	int *d_array;
	hipMalloc((void**) &d_array, sizeof(int)*(nRows*nCols));
    hipMemcpy(d_array, h_array,sizeof(int)*(nRows*nCols), hipMemcpyHostToDevice);

	//Execute on CPU
	double time1 = omp_get_wtime();
	digSize = 0;
	omp_set_dynamic(0);
    omp_set_num_threads(6);
	for (int i = 0; i < nRows; ++i)
	{	
		digSize = digSize + i;
		cpu_left(i, digSize, h_array, subsequence1, subsequence2);
	}

	for (int i = 1; i < 2; ++i)
	{	
		digSize = digSize + nRows - (i - 1);
		cpu_mid(i, digSize, h_array, subsequence1, subsequence2);
	}
	
	for (int i = 2; i < nRows; ++i)
	{	
		digSize = digSize + nRows - (i - 1);
		cpu_bottom(i, digSize, h_array, subsequence1, subsequence2);
	}

	cout << "\n";
	cout <<"Time on cpu: " <<omp_get_wtime() - time1 << "\n";
	cout << "Result on cpu: " << h_array [nRows*nCols - 1] << "\n";
	

	/*//Execute on GPU
	double time2 = omp_get_wtime();
	digSize = 0;
	for (int i = 0; i < nRows; ++i)
	{	
		digSize = digSize + i;
		configure_kernal(i-1);
		gpu_left<<<dim3(g,1,1), dim3(x,1,1)>>>(i, digSize, d_array, d_subsequence1, d_subsequence2);
	}

	for (int i = 1; i < 2; ++i)
	{	
		digSize = digSize + nRows - (i - 1);
		configure_kernal(nRows-i);
		gpu_mid<<<dim3(g,1,1), dim3(x,1,1)>>>(i, digSize, d_array, d_subsequence1, d_subsequence2);
	}
	
	for (int i = 2; i < nRows; ++i)
	{	
		digSize = digSize + nRows - (i - 1);
		configure_kernal(nRows-i);
		gpu_bottom<<<dim3(g,1,1), dim3(x,1,1)>>>(i, digSize, d_array, d_subsequence1, d_subsequence2);
	}

	cout << "\n";
	cout <<"Time on gpu: " <<omp_get_wtime() - time2 << "\n";

	
	//copyback h_array to cpu
	cudaMemcpy(h_array, d_array,sizeof(int)*nRows*nCols, cudaMemcpyDeviceToHost);
	cout << "Result on gpu: "<< h_array [nRows*nCols - 1] << "\n";
*/
	//Execute on Hybrid (CPU + GPU) : Handover

	/*double time3 = omp_get_wtime();
	digSize = 0;
	int i;
	for (i = 0; i < CUTOFF_HANDOVER; ++i)
	{	
		digSize = digSize + i;
		cpu_left(i, digSize, h_array, subsequence1, subsequence2);
	}
	//int locate = digSize - (2*i -1);
	//cudaMemcpy(d_array + locate, h_array + locate ,sizeof(int) * 2*i, cudaMemcpyHostToDevice);
	for (i = CUTOFF_HANDOVER; i < nRows; ++i)
	{	
		digSize = digSize + i;
		configure_kernal(i-1);
		gpu_left<<<dim3(g,1,1), dim3(x,1,1)>>>(i, digSize, d_array, d_subsequence1, d_subsequence2);
	}

	for (i = 1; i < 2; ++i)
	{	
		digSize = digSize + nRows - (i - 1);
		configure_kernal(nRows-i);
		gpu_mid<<<dim3(g,1,1), dim3(x,1,1)>>>(i, digSize, d_array, d_subsequence1, d_subsequence2);
	}
	
	for (i = 2; i < nRows - CUTOFF_HANDOVER; ++i)
	{	
		digSize = digSize + nRows - (i - 1);
		configure_kernal(nRows-i);
		gpu_bottom<<<dim3(g,1,1), dim3(x,1,1)>>>(i, digSize, d_array, d_subsequence1, d_subsequence2);
	}
	//locate = digSize - 2*(nRows - (i - 1));
	//cudaMemcpy(h_array + locate, d_array + locate ,sizeof(int) * 2*(nRows-i), cudaMemcpyDeviceToHost);
	for (i = nRows - CUTOFF_HANDOVER; i < nRows; ++i)
	{	
		digSize = digSize + nRows - (i - 1);
		cpu_bottom(i, digSize, h_array, subsequence1, subsequence2);
	}

	cout << "\n";
	cout <<"Time on Hybrid: " <<omp_get_wtime() - time3 << "\n";
	cout << "Result on Hybrid: "<< h_array [nRows*nCols - 1] << "\n";*/


	//convert into 2d matrix : in the original order i.e. row order


	digSize = 0;
	for (int i = 0; i < nRows; ++i)
	{	
		digSize = digSize + i;
		for (int j = 0; j <= i; ++j)
		{
			arrayOrg[i - j][j] = h_array[digSize + j];
		}
		
	}
	
	for (int i = 1; i < nRows; ++i)
	{	
		digSize = digSize + nRows - (i - 1);
		for (int j = 0; j < (nRows - i); ++j)
		{
			arrayOrg[nRows - j - 1][i+j] = h_array[digSize + j] ;
		}
	}
	//Access the resultant matrix or write to file
	
	ofstream myfile ("output_s.txt");
	for (int i = 0; i < nRows; ++i)
	{	
		for (int j = 0; j < nCols; ++j)
		{
			myfile << arrayOrg[i][j] << "\t";
			
		}
		myfile << "\n";
	}

	cout << "\n";
	/*ofstream myfile ("output_s.txt");
	for (int i = nRows*nCols -100; i < nRows*nCols; ++i)
	{
		myfile << h_array[i] << "\t";
	}*/
	
	
	return 0;
}

void cpu_left (int i, int digSize, int *h_array, char *subsequence1, char *subsequence2)
{
		//#pragma omp parallel for
		for (int j = 1; j <= i-1; ++j)
		{
			if (subsequence1 [i - j] == subsequence2 [j])
			{
				Z = 1 + B1;
			}
			else
			{
				( A1 > C1 ? Z = A1 : Z = C1 );
			}
		}
}
void cpu_mid(int i, int digSize, int *h_array, char *subsequence1, char *subsequence2)
{
		//#pragma omp parallel for
		for (int j = 0; j < (nRows - i); ++j)
		{
			if (subsequence1 [nRows - j - 1] == subsequence2 [i + j])
			{
				Z = 1 + B2ex;
			}
			else
			{
				( A2 > C2 ? Z = A2 : Z = C2 );
			}
		}
}
void cpu_bottom(int i, int digSize, int *h_array, char *subsequence1, char *subsequence2)
{
	//#pragma omp parallel for
	for (int j = 0; j < (nRows - i); ++j)
		{
			if (subsequence1 [nRows - j - 1] == subsequence2 [i + j])
			{
				Z = 1 + B2;
			}
			else
			{
				( A2 > C2 ? Z = A2 : Z = C2 );
			}
		}

}

__global__ void gpu_left (int i, int digSize, int *d_array, char *d_subsequence1, char *d_subsequence2)
{
		long j=blockIdx.x *blockDim.x + threadIdx.x + 1;
   		if (j > i-1)
      	{

      	}
		else
		{
			if (d_subsequence1 [i - j] == d_subsequence2 [j])
			{
				d_Z = 1 + d_B1;
			}
			else
			{
				( d_A1 > d_C1 ? d_Z = d_A1 : d_Z = d_C1 );
			}
		}
}
__global__ void gpu_mid(int i, int digSize, int *d_array, char *d_subsequence1, char *d_subsequence2)
{
		long j=blockIdx.x *blockDim.x + threadIdx.x ;
   		if (j >= (nRows - i))
      	{

      	}
		else
		{
			if (d_subsequence1 [nRows - j - 1] == d_subsequence2 [i + j])
			{
				d_Z = 1 + d_B2ex;
			}
			else
			{
				( d_A2 > d_C2 ? d_Z = d_A2 : d_Z = d_C2 );
			}
		}
}
__global__ void gpu_bottom(int i, int digSize, int *d_array, char *d_subsequence1, char *d_subsequence2)
{
		long j=blockIdx.x *blockDim.x + threadIdx.x ;
   		if (j >= (nRows - i))
      	{

      	}
		else
		{
			if (d_subsequence1 [nRows - j - 1] == d_subsequence2 [i + j])
			{
				d_Z = 1 + d_B2;
			}
			else
			{
				( d_A2 > d_C2 ? d_Z = d_A2 : d_Z = d_C2 );
			}
		}

}


void configure_kernal(long numberOfThreadsRequired)
{
   if (numberOfThreadsRequired <= BLOCK_SIZE)
      {
      	 g = 1; x = numberOfThreadsRequired ;
      }
   else
      {
         g = (numberOfThreadsRequired / BLOCK_SIZE)+1; x = BLOCK_SIZE;
      }

}