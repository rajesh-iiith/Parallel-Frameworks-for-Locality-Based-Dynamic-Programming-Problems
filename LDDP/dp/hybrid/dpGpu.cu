#include "hip/hip_runtime.h"

#include "headers/myHeaders.h"

using namespace std;


int main(int argc, char const *argv[])
{	
	
	
	// Original array is 2d array : input image / matrix / DP table
	int **arrayOrg; 
	arrayOrg=new int *[nRows];
	for(int z=0 ; z<nRows ; z++)
	{
		arrayOrg[z]=new int[nCols];
	}

	// Load external reseorces. e.g. subsequences in case of LCS
	//1. To CPU

	 char *subsequence1 = new char[nRows];
	 for (int i = 1; i < nRows; ++i)
	 {
	 	subsequence1 [i] = rand()%4 + 67;
	 }
	 char *subsequence2 = new char[nCols];
	 for (int i = 1; i < nRows; ++i)
	 {
	 	subsequence2 [i] = rand()%4 + 67;
	 }

	 //2. To GPU

	 char *d_subsequence1;
     hipMalloc((void**) &d_subsequence1, sizeof(char)*nRows);
     hipMemcpy(d_subsequence1, subsequence1,sizeof(char)*nRows, hipMemcpyHostToDevice);
     char *d_subsequence2;
     hipMalloc((void**) &d_subsequence2, sizeof(char)*nCols);
     hipMemcpy(d_subsequence2, subsequence2,sizeof(char)*nCols, hipMemcpyHostToDevice);

	// initialiation : not required if we are inputting the image : can be replaced by input code
	for (int i = 0; i < nRows; ++i)
	{	
		for (int j = 0; j < nCols; ++j)
		{
			arrayOrg[i][j] = 0;
		}
	}

	// memory coalescing : change to 1D array

	int *h_array = (int*)calloc(nRows*nCols, sizeof(int));
	
	int digSize = 0;
	for (int i = 0; i < nRows; ++i)
	{	
		digSize = digSize + i;
		for (int j = 0; j <= i; ++j)
		{
			h_array[digSize + j] = arrayOrg[i - j][j];
		}
		
	}
	
	for (int i = 1; i < nRows; ++i)
	{	
		digSize = digSize + nRows - (i - 1);
		for (int j = 0; j < (nRows - i); ++j)
		{
			h_array[digSize + j] = arrayOrg[nRows - j - 1][i+j];
		}
	}

	// Load main resource (DP table/ Image i.e. h_array) to GPU
	int *d_array;
	hipMalloc((void**) &d_array, sizeof(int)*(nRows*nCols));
    hipMemcpy(d_array, h_array,sizeof(int)*(nRows*nCols), hipMemcpyHostToDevice);


	//Execute on GPU
	struct timeval start, end;
 	gettimeofday(&start, NULL);

	double time1 = omp_get_wtime();

	digSize = 0;
	for (int i = 0; i < nRows; ++i)
	{	
		digSize = digSize + i;
		configure_kernal(i-1);
		gpu_left<<<dim3(g,1,1), dim3(x,1,1)>>>(i, digSize, d_array, d_subsequence1, d_subsequence2);
	}

	for (int i = 1; i < 2; ++i)
	{	
		digSize = digSize + nRows - (i - 1);
		configure_kernal(nRows-i);
		gpu_mid<<<dim3(g,1,1), dim3(x,1,1)>>>(i, digSize, d_array, d_subsequence1, d_subsequence2);
	}
	
	for (int i = 2; i < nRows; ++i)
	{	
		digSize = digSize + nRows - (i - 1);
		configure_kernal(nRows-i);
		gpu_bottom<<<dim3(g,1,1), dim3(x,1,1)>>>(i, digSize, d_array, d_subsequence1, d_subsequence2);
	}

	cout << "\n";
	gettimeofday(&end, NULL);
  	double run_time = ((end.tv_sec - start.tv_sec)*1000 + (end.tv_usec - start.tv_usec)/1000.0);
	printf("Time (gpu): %.3lf\n", run_time);
	//cout << "Time (gpu): "<< (omp_get_wtime() - time1) * 1000 << "\n";

	
	//copyback h_array to cpu
	hipMemcpy(h_array, d_array,sizeof(int)*nRows*nCols, hipMemcpyDeviceToHost);
	//cout << "Result on gpu: "<< h_array [nRows*nCols - 1] << "\n";


/*	//convert into 2d matrix : in the original order i.e. row order


	digSize = 0;
	for (int i = 0; i < nRows; ++i)
	{	
		digSize = digSize + i;
		for (int j = 0; j <= i; ++j)
		{
			arrayOrg[i - j][j] = h_array[digSize + j];
		}
		
	}
	
	for (int i = 1; i < nRows; ++i)
	{	
		digSize = digSize + nRows - (i - 1);
		for (int j = 0; j < (nRows - i); ++j)
		{
			arrayOrg[nRows - j - 1][i+j] = h_array[digSize + j] ;
		}
	}
	//Access the resultant matrix or write to file
	
	ofstream myfile ("output_s.txt");
	for (int i = 0; i < nRows; ++i)
	{	
		for (int j = 0; j < nCols; ++j)
		{
			myfile << arrayOrg[i][j] << "\t";
			
		}
		myfile << "\n";
	}

	cout << "\n";
*/	
	return 0;
}





__global__ void gpu_left (int i, int digSize, int *d_array, char *d_subsequence1, char *d_subsequence2)
{
		long j=blockIdx.x *blockDim.x + threadIdx.x + 1;
   		if (j > i-1)
      	{

      	}
		else
		{
			if (d_subsequence1 [i - j] == d_subsequence2 [j])
			{
				d_Z = 1 + d_B1;
			}
			else
			{
				( d_A1 > d_C1 ? d_Z = d_A1 : d_Z = d_C1 );
			}
		}
}
__global__ void gpu_mid(int i, int digSize, int *d_array, char *d_subsequence1, char *d_subsequence2)
{
		long j=blockIdx.x *blockDim.x + threadIdx.x ;
   		if (j >= (nRows - i))
      	{

      	}
		else
		{
			if (d_subsequence1 [nRows - j - 1] == d_subsequence2 [i + j])
			{
				d_Z = 1 + d_B2ex;
			}
			else
			{
				( d_A2 > d_C2 ? d_Z = d_A2 : d_Z = d_C2 );
			}
		}
}
__global__ void gpu_bottom(int i, int digSize, int *d_array, char *d_subsequence1, char *d_subsequence2)
{
		long j=blockIdx.x *blockDim.x + threadIdx.x ;
   		if (j >= (nRows - i))
      	{

      	}
		else
		{
			if (d_subsequence1 [nRows - j - 1] == d_subsequence2 [i + j])
			{
				d_Z = 1 + d_B2;
			}
			else
			{
				( d_A2 > d_C2 ? d_Z = d_A2 : d_Z = d_C2 );
			}
		}

}


void configure_kernal(long numberOfThreadsRequired)
{
   if (numberOfThreadsRequired <= BLOCK_SIZE)
      {
      	 g = 1; x = numberOfThreadsRequired ;
      }
   else
      {
         g = (numberOfThreadsRequired / BLOCK_SIZE)+1; x = BLOCK_SIZE;
      }

}