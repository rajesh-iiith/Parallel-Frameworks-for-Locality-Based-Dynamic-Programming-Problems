
#include "headers/myHeaders.h"

using namespace std;


int main(int argc, char const *argv[])
{	
	
	
	// Original array is 2d array : input image / matrix / DP table
	int **arrayOrg; 
	arrayOrg=new int *[nRows];
	for(int z=0 ; z<nRows ; z++)
	{
		arrayOrg[z]=new int[nCols];
	}

	// Load external reseorces. e.g. subsequences in case of LCS
	//1. To CPU

	 char *subsequence1 = new char[nRows];
	 for (int i = 1; i < nRows; ++i)
	 {
	 	subsequence1 [i] = rand()%4 + 67;
	 }
	 char *subsequence2 = new char[nCols];
	 for (int i = 1; i < nRows; ++i)
	 {
	 	subsequence2 [i] = rand()%4 + 67;
	 }

	 //2. To GPU

	 char *d_subsequence1;
     hipMalloc((void**) &d_subsequence1, sizeof(char)*nRows);
     hipMemcpy(d_subsequence1, subsequence1,sizeof(char)*nRows, hipMemcpyHostToDevice);
     char *d_subsequence2;
     hipMalloc((void**) &d_subsequence2, sizeof(char)*nCols);
     hipMemcpy(d_subsequence2, subsequence2,sizeof(char)*nCols, hipMemcpyHostToDevice);

	// initialiation : not required if we are inputting the image : can be replaced by input code
	for (int i = 0; i < nRows; ++i)
	{	
		for (int j = 0; j < nCols; ++j)
		{
			arrayOrg[i][j] = 0;
		}
	}

	// memory coalescing : change to 1D array

	int *h_array = (int*)calloc(nRows*nCols, sizeof(int));
	
	int digSize = 0;
	for (int i = 0; i < nRows; ++i)
	{	
		digSize = digSize + i;
		for (int j = 0; j <= i; ++j)
		{
			h_array[digSize + j] = arrayOrg[i - j][j];
		}
		
	}
	
	for (int i = 1; i < nRows; ++i)
	{	
		digSize = digSize + nRows - (i - 1);
		for (int j = 0; j < (nRows - i); ++j)
		{
			h_array[digSize + j] = arrayOrg[nRows - j - 1][i+j];
		}
	}

	// Load main resource (DP table/ Image i.e. h_array) to GPU
	int *d_array;
	hipMalloc((void**) &d_array, sizeof(int)*(nRows*nCols));
    hipMemcpy(d_array, h_array,sizeof(int)*(nRows*nCols), hipMemcpyHostToDevice);

	//Execute on CPU
	struct timeval start, end;
 	gettimeofday(&start, NULL);
	double time1 = omp_get_wtime();

	digSize = 0;
	omp_set_dynamic(0);
    omp_set_num_threads(6);
	for (int i = 0; i < nRows; ++i)
	{	
		digSize = digSize + i;
		cpu_left(i, digSize, h_array, subsequence1, subsequence2);
	}

	for (int i = 1; i < 2; ++i)
	{	
		digSize = digSize + nRows - (i - 1);
		cpu_mid(i, digSize, h_array, subsequence1, subsequence2);
	}
	
	for (int i = 2; i < nRows; ++i)
	{	
		digSize = digSize + nRows - (i - 1);
		cpu_bottom(i, digSize, h_array, subsequence1, subsequence2);
	}

	cout << "\n";
	gettimeofday(&end, NULL);
  	double run_time = ((end.tv_sec - start.tv_sec)*1000 + (end.tv_usec - start.tv_usec)/1000.0);
	printf("Time (cpu): %.3lf\n", run_time);
	//cout <<"Time (cpu): " <<(omp_get_wtime() - time1)*1000<< "\n";
	//cout << "Result on cpu: " << h_array [nRows*nCols - 1] << "\n";
	


	
	//convert into 2d matrix : in the original order i.e. row order
/*	digSize = 0;
	for (int i = 0; i < nRows; ++i)
	{	
		digSize = digSize + i;
		for (int j = 0; j <= i; ++j)
		{
			arrayOrg[i - j][j] = h_array[digSize + j];
		}
		
	}
	
	for (int i = 1; i < nRows; ++i)
	{	
		digSize = digSize + nRows - (i - 1);
		for (int j = 0; j < (nRows - i); ++j)
		{
			arrayOrg[nRows - j - 1][i+j] = h_array[digSize + j] ;
		}
	}
	//Access the resultant matrix or write to file
	
	ofstream myfile ("../output_s.txt");
	for (int i = 0; i < nRows; ++i)
	{	
		for (int j = 0; j < nCols; ++j)
		{
			myfile << arrayOrg[i][j] << "\t";
			
		}
		myfile << "\n";
	}

	cout << "\n";*/
	
	/*ofstream myfile ("output_s.txt");
	for (int i = nRows*nCols -100; i < nRows*nCols; ++i)
	{
		myfile << h_array[i] << "\t";
	}*/
	
	
	return 0;
}

void cpu_left (int i, int digSize, int *h_array, char *subsequence1, char *subsequence2)
{
		#pragma omp parallel for
		for (int j = 1; j <= i-1; ++j)
		{
			if (subsequence1 [i - j] == subsequence2 [j])
			{
				Z = 1 + B1;
			}
			else
			{
				( A1 > C1 ? Z = A1 : Z = C1 );
			}
		}
}
void cpu_mid(int i, int digSize, int *h_array, char *subsequence1, char *subsequence2)
{
		#pragma omp parallel for
		for (int j = 0; j < (nRows - i); ++j)
		{
			if (subsequence1 [nRows - j - 1] == subsequence2 [i + j])
			{
				Z = 1 + B2ex;
			}
			else
			{
				( A2 > C2 ? Z = A2 : Z = C2 );
			}
		}
}
void cpu_bottom(int i, int digSize, int *h_array, char *subsequence1, char *subsequence2)
{
	#pragma omp parallel for
	for (int j = 0; j < (nRows - i); ++j)
		{
			if (subsequence1 [nRows - j - 1] == subsequence2 [i + j])
			{
				Z = 1 + B2;
			}
			else
			{
				( A2 > C2 ? Z = A2 : Z = C2 );
			}
		}

}
