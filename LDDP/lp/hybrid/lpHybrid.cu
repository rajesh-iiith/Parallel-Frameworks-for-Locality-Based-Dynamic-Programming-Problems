#include "hip/hip_runtime.h"
#include "headers/myHeaders.h"

//using statements
using namespace std;


int main(int argc, char const *argv[])
{	
	omp_set_nested(1);
    omp_set_dynamic(1);
	// Original array is 2d array : input image / matrix / DP table
	int **arrayOrg; 
	arrayOrg=new int *[nRows];
	for(int z=0 ; z<nRows ; z++)
	{
		arrayOrg[z]=new int[nCols];
	}

	// initialiation : not required if we are inputting the image : can be replaced by input code
	for (int i = 0; i < nRows; ++i)
	{	
		for (int j = 0; j < nCols; ++j)
		{
			arrayOrg[i][j] = 1;
		}
	}

	// memory coalescing : Store in cloumn major order (since vertical parallelism), change to 1D array

	//thrust::host_vector<int> h_array(nRows * nCols);
	int *h_array = (int*)calloc(nRows*nCols, sizeof(int));
	int m = nRows;
	int n = nCols;
	int count = 0;
	int smallerDim;
	(nRows > nCols ? smallerDim = nCols : smallerDim = nRows );

	for (int i = 0; i < smallerDim; ++i)
	{	
		for (int j = 0; j < n; ++j)
		{
			h_array[count + j] = arrayOrg[i][j+i];
		}
		for (int j = 0; j < m-1; ++j)
		{
			h_array[count + n + j] = arrayOrg[j+1+i][i];
		}
		count =  (m + n - 1) + count;
		m = m - 1;
		n = n - 1;
		
	}

	/*// operations on cpu
	omp_set_num_threads(6);
	count = nRows + nCols -1;
	m = nRows - 1;
	n = nCols - 1;
	int oldCount = 0;
	double time1 = omp_get_wtime();
	if (n > m)
	{
		for (int i = 1; i < smallerDim; ++i)
		{	
			update_array_one_cpu(m, n, count, oldCount, h_array);
			oldCount = count;
			count =  (m + n - 1) + count;
			m = m - 1;
			n = n - 1;
			
		}
	}
	else
	{
		for (int i = 1; i < smallerDim; ++i)
		{	
			update_array_two_cpu(m, n, count, oldCount, h_array);
			oldCount = count;
			count =  (m + n - 1) + count;
			m = m - 1;
			n = n - 1;
			
		}
	}
	
	cout << "\n";
	cout << omp_get_wtime() - time1 << endl;*/

	//copy the h_array to gpu
	int *d_array;
	hipMalloc((void**) &d_array, sizeof(int)*(nRows*nCols));
    hipMemcpy(d_array, h_array,sizeof(int)*(nRows*nCols), hipMemcpyHostToDevice);

	
	//call kernel
	//omp_set_num_threads(1);
	count = nRows + nCols -1;
	m = nRows - 1;
	n = nCols - 1;
	int oldCount = 0;
	double time2 = omp_get_wtime();

	if (n > m)
	{
		for (int i = 1; i < (smallerDim - cpu_work_factor); ++i)
		{	
			configure_kernal(n);
			update_array_one_gpu<<<dim3(g,1,1), dim3(x,1,1)>>>(m, n, i, n, count, oldCount, d_array);
			oldCount = count;
			count =  (m + n - 1) + count;
			m = m - 1;
			n = n - 1;
			
		}
		//hipMemcpy(h_array, d_array,sizeof(int)*nRows*nCols, hipMemcpyDeviceToHost);
		for (int i = (smallerDim - cpu_work_factor); i < smallerDim; ++i)
		{	
			update_array_one_cpu(m, n, count, oldCount, h_array);
			oldCount = count;
			count =  (m + n - 1) + count;
			m = m - 1;
			n = n - 1;
			
		}
	}

	else
	{
		for (int i = 1; i < (smallerDim - cpu_work_factor); ++i)
		{	
			configure_kernal(m);
			update_array_two_gpu<<<dim3(g,1,1), dim3(x,1,1)>>>(m, n, i, m, count, oldCount, d_array);
			oldCount = count;
			count =  (m + n - 1) + count;
			m = m - 1;
			n = n - 1;
			
		}
		//hipMemcpy(h_array, d_array,sizeof(int)*nRows*nCols, hipMemcpyDeviceToHost);
		for (int i = (smallerDim - cpu_work_factor); i < smallerDim; ++i)
		{	
			update_array_two_cpu(m, n, count, oldCount, h_array);
			oldCount = count;
			count =  (m + n - 1) + count;
			m = m - 1;
			n = n - 1;
			
		}
	}
	
	cout << "\n";
	
	cout << (omp_get_wtime() - time2)*1000 <<endl;
	//copyback to cpu
	//hipMemcpy(h_array, d_array,sizeof(int)*nRows*nCols, hipMemcpyDeviceToHost);


	//display original 2d array : for debugging
	/*for (int i = 0; i < nRows; ++i)
	{
		for (int j = 0; j < nCols; ++j)
		{
			cout << arrayOrg[i][j] <<" " ;
		}
		cout << "\n";
	}*/

	/*cout << "\n";
	//display original 1d array : for debugging
	for (int i = 0; i < nRows*nCols; ++i)
	{
		cout << h_array[i] << " ";  
	}*/

	//convert into 2d matrix : in the original order 
	m = nRows;
	n = nCols;
	count = 0;
	
	for (int i = 0; i < smallerDim; ++i)
	{	
		for (int j = 0; j < n; ++j)
		{
			arrayOrg[i][j+i] = h_array[count + j];
		}
		for (int j = 0; j < m-1; ++j)
		{
			arrayOrg[j+1+i][i] = h_array[count + n + j];
		}
		count =  (m + n - 1) + count;
		m = m - 1;
		n = n - 1;
		
	}

	//Access the resultant matrix

	/*for (int i = 0; i < nRows; ++i)
	{	
		for (int j = 0; j < nCols; ++j)
		{
			cout << arrayOrg[i][j] ;
			//myfile << arrayOrg[i][j] << "\t";
		}
		//myfile << "\n";
		cout << "\n" ;
	}*/
	
	return 0;
}

__global__ void update_array_one_gpu(int m, int n, int i, int numberOfThreadsRequired,int count, int oldCount, int *d_array )
{
   long j=blockIdx.x *blockDim.x + threadIdx.x;

   if (j> numberOfThreadsRequired)
      {}
   else
   {
       	 GPU_Expression_1;
	     if (j < (m - 1) )
			{
				GPU_Expression_2;
			}   
   }
}


__global__ void update_array_two_gpu(int m, int n, int i, int numberOfThreadsRequired, int count, int oldCount, int *d_array )
{
   long j=blockIdx.x *blockDim.x + threadIdx.x;

   if (j> numberOfThreadsRequired)
      {}
   else
   {
       	    GPU_Expression_2;	
			if (j < n)
			{
				GPU_Expression_1;
			} 
   }
}
void update_array_one_cpu(int m, int n, int count, int oldCount, int *h_array)
{
	//#pragma omp parallel for
		for (int j = 0; j < n; ++j)
		{
			CPU_Expression_1;
			if (j < (m - 1) )
			{
				CPU_Expression_2;
			}
		}	
}
void update_array_two_cpu(int m, int n, int count, int oldCount, int *h_array)
{
	//#pragma omp parallel for
		for (int j = 0; j < m; ++j)
		{
			CPU_Expression_2;	
			if (j < n)
			{
				CPU_Expression_1;
			}
		}
}
void configure_kernal(long numberOfThreadsRequired)
{
   if (numberOfThreadsRequired <=1024)
      {x=numberOfThreadsRequired ; g=1;}
   else
      {
         g= (numberOfThreadsRequired/1024)+1; x= 1024;
      }

}