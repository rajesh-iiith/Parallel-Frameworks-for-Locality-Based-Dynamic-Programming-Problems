#include "hip/hip_runtime.h"
#include "headers/myHeaders.h"

//using statements
using namespace std;

int main(int argc, char const *argv[])
{ 
  // Original array is 2d array : input image / matrix / DP table 
  arrayOrg=new int *[nRows];
  for(int z=0 ; z<nRows ; z++)
  {
    arrayOrg[z]=new int[nCols];
  }

  //flatten the array to 1d array
  h_array = (int*)calloc(nRows*nCols, sizeof(int));
  for (int i = 0; i < nRows; ++i)
  {
    for (int j = 0; j < nCols; ++j)
    {
     //h_array[i*nCols + j] = arrayOrg [i][j];
      h_array[i*nCols + j] = rand() % 10;
    }
  }
  //free arrayOrg
  for(int z=0 ; z<nRows ; z++)
  {
    free(arrayOrg[z]);
  }
  free(arrayOrg);

  //load h_array to GPU
  // Load main resource (DP table/ Image i.e. h_array) to GPU
  int *d_array;
  hipMalloc((void**) &d_array, sizeof(int)*(nRows*nCols));
  hipMemcpy(d_array, h_array,sizeof(int)*(nRows*nCols), hipMemcpyHostToDevice);



  // initialiation : not required if we are inputting the image : can be replaced by input code
  /*for (int i = 0; i < nRows; ++i)
  { 
    for (int j = 0; j < nCols; ++j)
    {
      h_array[i*nCols + j] = rand() % 10;
    }
  }*/


 double time1 = omp_get_wtime();
 int x_start = 1;
 int y_start = 1;
 //omp_set_nested(1);
 for (int i = 1; i <= (nRows/tileLength) ; ++i)
  {
      operate_on_block_gpu <<<dim3(i,1,1), dim3(tileLength,1,1)>>> (i, d_array, x_start, y_start);
      x_start = x_start + tileLength;
  } 

    x_start = x_start - tileLength;
    y_start = y_start + tileLength;
    for (int i = (nRows/tileLength)-1; i >= 1 ; --i)
  {
      operate_on_block_gpu <<<dim3(i,1,1), dim3(tileLength,1,1)>>> (i, d_array, x_start, y_start);
      y_start = y_start + tileLength;
  } 

  cout << "Time (Blocked): " << (omp_get_wtime() - time1)*1000 << "\n";
  hipMemcpy(h_array , d_array ,sizeof(int) * (nRows*nCols), hipMemcpyDeviceToHost);

  //Access the resultant matrix or write to file
  
 /* ofstream myfile ("../output_t.txt");
  for (int i = 0; i < nRows; ++i)
  { 
    for (int j = 0; j < nCols; ++j)
    {
      myfile << h_array[i*nCols + j] << "\t";
      
    }
    myfile << "\n";
  }*/
  
  cout << "Result(Blocked): "<< h_array[nRows*nCols -1] <<"\n";
  return 0;
}


__global__ void operate_on_block_gpu (int i, int *d_array, int x_start, int y_start)
{
      //long tid=blockIdx.x *blockDim.x + threadIdx.x;
      
       
          int x_my_block = x_start - (blockIdx.x) * tileLength;
          int y_my_block = y_start + (blockIdx.x) * tileLength;
        

        //operate_on_block (x_start, y_start, subsequence1, subsequence2);

                      int x_start_local = x_my_block;
                      int y_start_local = y_my_block;
                        for (int i = 1; i <= tileLength; ++i)
                        {
                          int x_my = x_start_local + (i-1)*1;
                          
                          for (int j = 1; j <= tileLength; ++j)
                          {
                            int y_my = y_start_local + (j-1)*1;
                ExpressionGPU;
                          }
                            
                            
                        }
    
}

