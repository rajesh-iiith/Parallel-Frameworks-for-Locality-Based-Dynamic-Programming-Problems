//include statements

#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include "omp.h"
#include <fstream>

//define statements
#define nRows 16385
#define nCols 16385
#define tileLength 32
// 128 is best block size in case of lcs (on the basis of experiments on 10k * 10k)
#define BLOCK_SIZE 128
#define CUTOFF_HANDOVER 500
#define CUTOFF_HYBRID 0

#define ExpressionGPU ( (d_B <= d_C) ? (d_Z = d_B + d_Z) : (d_Z = d_C + d_Z))
#define ExpressionCPU ( (B <= C) ? (Z = B + Z) : (Z = C + Z))


#define A h_array[ (x_my)*nCols + (y_my - 1) ]
#define B h_array[ (x_my - 1)*nCols + (y_my - 1) ]
#define C h_array[ (x_my - 1)*nCols + (y_my) ]
#define Z h_array[ (x_my)*nCols + (y_my) ]

#define d_A d_array[ (x_my)*nCols + (y_my - 1) ]
#define d_B d_array[ (x_my - 1)*nCols + (y_my - 1) ]
#define d_C d_array[ (x_my - 1)*nCols + (y_my) ]
#define d_Z d_array[ (x_my)*nCols + (y_my) ]



//using statements
using namespace std;

//global variables
int x,g;
int **arrayOrg;
int *h_array;
//CPU declarations
void operate_on_block_cpu (int i, int x_start, int y_start);
__global__ void operate_on_block_gpu (int i, int *d_array, int x_start, int y_start);

int main(int argc, char const *argv[])
{ 
  // Original array is 2d array : input image / matrix / DP table 
  arrayOrg=new int *[nRows];
  for(int z=0 ; z<nRows ; z++)
  {
    arrayOrg[z]=new int[nCols];
  }

  //flatten the array to 1d array
  h_array = (int*)calloc(nRows*nCols, sizeof(int));
  for (int i = 0; i < nRows; ++i)
  {
    for (int j = 0; j < nCols; ++j)
    {
     //h_array[i*nCols + j] = arrayOrg [i][j];
      h_array[i*nCols + j] = rand() % 10;
    }
  }
  //free arrayOrg
  for(int z=0 ; z<nRows ; z++)
  {
    free(arrayOrg[z]);
  }
  free(arrayOrg);

  //load h_array to GPU
  // Load main resource (DP table/ Image i.e. h_array) to GPU
  int *d_array;
  hipMalloc((void**) &d_array, sizeof(int)*(nRows*nCols));
  hipMemcpy(d_array, h_array,sizeof(int)*(nRows*nCols), hipMemcpyHostToDevice);



  // initialiation : not required if we are inputting the image : can be replaced by input code
  /*for (int i = 0; i < nRows; ++i)
  { 
    for (int j = 0; j < nCols; ++j)
    {
      h_array[i*nCols + j] = rand() % 10;
    }
  }*/

/*for (int i = 0; i < nRows; ++i)
  { 
    for (int j = 0; j < nCols; ++j)
    {
      cout << h_array[i*nCols + j] << "\t";
      
    }
    cout << "\n";
  }*/
 double time1 = omp_get_wtime();
 int x_start = 1;
 int y_start = 1;
 //omp_set_nested(1);
 for (int i = 1; i <= (nRows/tileLength) ; ++i)
  {
    
    //operate_on_block_gpu <<<dim3(i,1,1), dim3(tileLength,1,1)>>> (i, d_array, x_start, y_start);
    operate_on_block_cpu(i, x_start, y_start);
      x_start = x_start + tileLength;
  } 

    x_start = x_start - tileLength;
    y_start = y_start + tileLength;
    for (int i = (nRows/tileLength)-1; i >= 1 ; --i)
  {
    
      //operate_on_block_gpu <<<dim3(i,1,1), dim3(tileLength,1,1)>>> (i, d_array, x_start, y_start);
      operate_on_block_cpu(i, x_start, y_start);
      y_start = y_start + tileLength;
  } 

  cout << "Time (Blocked): " << (omp_get_wtime() - time1)*1000 << "\n";
  //cudaMemcpy(h_array , d_array ,sizeof(int) * (nRows*nCols), cudaMemcpyDeviceToHost);

  //Access the resultant matrix or write to file
  
  //ofstream myfile ("output.txt");
  /*for (int i = 0; i < nRows; ++i)
  { 
    for (int j = 0; j < nCols; ++j)
    {
      cout << h_array[i*nCols + j] << "\t";
      
    }
    cout << "\n";
  }

  cout << "\n";*/
  
  cout << "Result(Blocked): "<< h_array[nRows*nCols -1] <<"\n";
  return 0;
}


__global__ void operate_on_block_gpu (int i, int *d_array, int x_start, int y_start)
{
      //long tid=blockIdx.x *blockDim.x + threadIdx.x;
      
       
          int x_my_block = x_start - (blockIdx.x) * tileLength;
          int y_my_block = y_start + (blockIdx.x) * tileLength;
        

        //operate_on_block (x_start, y_start, subsequence1, subsequence2);

                      int x_start_local = x_my_block;
                      int y_start_local = y_my_block;
                        for (int i = 1; i <= tileLength; ++i)
                        {
                          int x_my = x_start_local + (i-1)*1;
                          
                          //#pragma omp parallel for
                          for (int j = 1; j <= tileLength; ++j)
                          {
                            int y_my = y_start_local + (j-1)*1;
                ExpressionGPU;
                          }
                            
                        }
    
}

void operate_on_block_cpu (int i, int x_start, int y_start)
{
      #pragma omp parallel for
      for (int j = 1; j <= i; ++j)
      {
        int x_my_block = x_start - (j-1)*tileLength;
        int y_my_block = y_start + (j-1)*tileLength;

        //operate_on_block (x_start, y_start, subsequence1, subsequence2);

                int x_start_local = x_my_block;
                      int y_start_local = y_my_block;
                        for (int i = 1; i <= tileLength; ++i)
                        {
                          int x_my = x_start_local + (i-1)*1;
                          
                          //#pragma omp parallel for
                          for (int j = 1; j <= tileLength; ++j)
                          {
                            int y_my = y_start_local + (j-1)*1;
              ExpressionCPU;
                          }
                            
                        }

          }
    
}

