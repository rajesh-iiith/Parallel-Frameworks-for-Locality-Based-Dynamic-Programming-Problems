//program for triangular tiling scheme for horizintal parallelism

//include statements

#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include "omp.h"
#include <fstream>
#include <iomanip>

//define statements
// 4097 8193 16385
#define nRows 16385
#define nCols 16385
#define tileLength 16
// 128 is best block size in case of lcs (on the basis of experiments on 10k * 10k)
#define BLOCK_SIZE 128
#define CUTOFF_HANDOVER 500
#define CUTOFF_HYBRID 0

//recheck D things
#define ExpressionCPU Z = (B + C + 2) / 2
#define ExpressionGPU d_Z = (d_B + d_C + 2) / 2 

#define A h_array[ (x_my)*nCols + (y_my - 1) ]
#define B h_array[ (x_my - 1)*nCols + (y_my - 1) ]
#define C h_array[ (x_my - 1)*nCols + (y_my) ]
#define D h_array[ (x_my - 1)*nCols + (y_my + 1) ]
#define Z h_array[ (x_my)*nCols + (y_my) ]

#define d_A d_array[ (x_my)*nCols + (y_my - 1) ]
#define d_B d_array[ (x_my - 1)*nCols + (y_my - 1) ]
#define d_C d_array[ (x_my - 1)*nCols + (y_my) ]
#define d_Z d_array[ (x_my)*nCols + (y_my) ]



//using statements
using namespace std;

//global variables
int x,g;
int **arrayOrg;
int *h_array;
//CPU declarations
void operate_on_valleys (int itr, int x_start, int y_start);
void operate_on_peaks (int itr, int x_start, int y_start);
void fix_boundary_tile_left(int itr, int x_start, int y_start);
void fix_boundary_tile_right(int itr, int x_start, int y_start);

__global__ void operate_on_valleys_gpu (int i, int *d_array, int x_start, int y_start);
__global__ void operate_on_peaks_gpu (int i, int *d_array, int x_start, int y_start);
__global__ void fix_boundary_tile_left_gpu(int itr, int *d_array, int x_start,int y_start);
__global__ void fix_boundary_tile_right_gpu(int itr, int *d_array, int x_start,int y_start);

int main(int argc, char const *argv[])
{ 
  // Original array is 2d array : input image / matrix / DP table 
  arrayOrg=new int *[nRows];
  for(int z=0 ; z<nRows ; z++)
  {
    arrayOrg[z]=new int[nCols];
  }

  //flatten the array to 1d array
  h_array = (int*)calloc(nRows*nCols, sizeof(int));
  for (int i = 0; i < nRows; ++i)
  {
    for (int j = 0; j < nCols; ++j)
    {
     h_array[i*nCols + j] = arrayOrg [i][j];
    }
  }
  //free arrayOrg
  for(int z=0 ; z<nRows ; z++)
  {
    free(arrayOrg[z]);
  }
  free(arrayOrg);


  // initialiation : not required if we are inputting the image : can be replaced by input code
  for (int i = 0; i < nRows; ++i)
  { 
    for (int j = 0; j < nCols; ++j)
    {
      h_array[i*nCols + j] = 0;
    }
  }

  //load h_array to GPU
  // Load main resource (DP table/ Image i.e. h_array) to GPU
  int *d_array;
  hipMalloc((void**) &d_array, sizeof(int)*(nRows*nCols));
  hipMemcpy(d_array, h_array,sizeof(int)*(nRows*nCols), hipMemcpyHostToDevice);

 double time1 = omp_get_wtime();

 int x_start = 1;
 int y_start = 1;
 //omp_set_nested(1);

 for (int itr = 1; itr <= (nRows-1)/tileLength; ++itr)
 {
       // valleys
        operate_on_valleys_gpu <<<dim3((nRows-1)/(2*tileLength),1,1), dim3(2*tileLength,1,1)>>> (itr, d_array, x_start, y_start);
        //operate_on_valleys (itr, x_start, y_start);
        y_start = (2 * tileLength);
        // peaks 
        operate_on_peaks_gpu <<<dim3((nRows-1)/(2*tileLength)-1,1,1), dim3(2*tileLength,1,1)>>> (itr, d_array, x_start, y_start);
        //operate_on_peaks(itr, x_start, y_start);
       
        //fix boundary triangles : can also use squares for ease
        y_start = 1;
        fix_boundary_tile_left_gpu<<<dim3(1,1,1), dim3(tileLength,1,1)>>>(itr, d_array, x_start, y_start);
        //fix_boundary_tile_left(itr, x_start, y_start);
        y_start = (nRows-1);
        fix_boundary_tile_right_gpu<<<dim3(1,1,1), dim3(tileLength,1,1)>>>(itr, d_array, x_start, y_start);
        //fix_boundary_tile_right(1, x_start, y_start);

        y_start = 1;
        x_start = x_start + tileLength;

  }
  cout << "Time (Blocked): " << omp_get_wtime() - time1 << "\n";
  hipMemcpy(h_array , d_array ,sizeof(int) * (nRows*nCols), hipMemcpyDeviceToHost);
  
  //Access the resultant matrix or write to file
  
  /*for (int i = 0; i < 50; ++i)
  { 
    for (int j = 0; j < 50; ++j)
    {
      cout << setfill('0') << setw(2) << h_array[i*nCols + j] << " ";
    }
    cout << "\n";
  }*/

  // cout << "\n";
  
  // for (int i = nRows-1; i < nRows; ++i)
  // { 
  //   for(int j = 0; j < nRows; ++j)
  //   {
  //     if(h_array[i*nCols + j] + 1 != h_array[i * nCols + j + 1])
  //     {
  //       printf("%d, %d\n", i, j);
  //       break;
  //     }
  //   }
  // }

  // for (int i = 0; i < nRows; ++i)
  // { 
  //   if(h_array[i*nCols + nRows-1] + 1 != h_array[(i+1) * nCols + nRows-1])
  //   {
  //     printf("%d\n", i);
  //     break;
  //   }
  // }

  cout << "\n";

  cout << "Result(triangular Blocked): "<< h_array[nRows*nCols -1] <<"\n";
  return 0;
}

__global__ void operate_on_valleys_gpu (int i, int *d_array, int x_start, int y_start)
{
    
        y_start = 1 + (blockIdx.x)*(tileLength *2);
        //y_start = 1 + (i-1)*(tileLength *2); 

            int x_my = x_start;
            int y_my = y_start;
            for (int j = tileLength; j >= 1; --j)
            {   
              int k = threadIdx.x + 1;
                
                if ( k <= (2*j) )
                {
                  y_my = y_my + (k-1);
                  ExpressionGPU;
                }
                
                x_my = x_my + 1;
                y_my = y_start +  tileLength - j + 1;
            }

   
}
__global__ void operate_on_peaks_gpu (int i, int *d_array, int x_start, int y_start)
{

              y_start = (2 * tileLength) + (blockIdx.x)*(tileLength *2);
            //y_start = (2 * tileLength) + (i-1)*(tileLength *2); 

              int x_my = x_start + 1;
              int y_my = y_start;
              
              for (int j = 2; j <= tileLength; ++j)
              {   
                  int k = threadIdx.x + 1;
                    if (k <= (2*(j-1)))
                    {
                      y_my = y_my + (k-1);
                      ExpressionGPU;
                    }

                  x_my = x_my + 1;
                  y_my = y_start - j + 1;
              }

}


__global__ void fix_boundary_tile_left_gpu(int itr, int *d_array, int x_start,int y_start)
{
      int x_my = x_start + 1;
      int y_my = y_start;

      for (int j = 2; j <= tileLength; ++j)
      {   
          int k = threadIdx.x + 1;
          if (k <= (j-1))
          {
            y_my = y_my + (k-1);
            ExpressionGPU;
          }
          x_my = x_my + 1;
          y_my = 1;
      }
}

__global__ void fix_boundary_tile_right_gpu(int itr, int *d_array, int x_start,int y_start)
{
      int x_my = x_start + 1;
      int y_my = y_start;

      for (int j = 2; j <= tileLength; ++j)
      {   
          int k = threadIdx.x + 1;
          if (k <= (j-1))
          {
            y_my = y_my - (k-1);
            ExpressionGPU;
          }
          x_my = x_my + 1;
          y_my = y_start;
      }
}

void operate_on_valleys (int itr, int x_start, int y_start)
{     
      #pragma omp parallel for private(y_start)
       for (int i = 1; i <= (nRows-1) / (tileLength * 2) ; ++i)
        { 
          y_start = 1 + (i-1)*(tileLength *2);  
            int x_my = x_start;
            int y_my = y_start;
            //cout << " test: "<< x_my << "," << y_my << "\n";
            for (int j = tileLength; j >= 1; --j)
            {   
                //#pragma omp parallel for
                for (int k = 1; k <= (2*j); ++k)
                {
                  //cout << " debug:: "<< x_my << "," << y_my << "\n";
                  ExpressionCPU;
                  y_my = y_my + 1;
                }
                
                x_my = x_my + 1;
                y_my = y_start +  tileLength - j + 1;
            }

    }
}

void operate_on_peaks (int itr, int x_start, int y_start)
{    

      #pragma omp parallel for private(y_start)
       for (int i = 1; i <= ((nRows-1) / (2*tileLength)) - 1; ++i)
        {   
            y_start = (2 * tileLength) + (i-1)*(tileLength *2);   
              int x_my = x_start + 1;
              int y_my = y_start;
              //cout << " test: "<< x_my << "," << y_my << "\n";
              for (int j = 2; j <= tileLength; ++j)
              {   
                  //#pragma omp parallel for
                  for (int k = 1; k <= (2*(j-1)); ++k)
                  {
                    //cout << " debug:: "<< x_my << "," << y_my << "\n";
                    ExpressionCPU;
                    y_my = y_my + 1;
                  }
                  
                  x_my = x_my + 1;
                  y_my = y_start - j + 1;
              }

        }
}

void fix_boundary_tile_left(int itr, int x_start, int y_start)
{ 
      int x_my = x_start + 1;
      int y_my = y_start;

      for (int j = 2; j <= tileLength; ++j)
      {
          //#pragma omp parallel for
          for (int k = 1; k <= (j-1); ++k)
          {
            //cout << " debug:: "<< x_my << "," << y_my << "\n";
            ExpressionCPU;
            y_my = y_my + 1;
          }
          x_my = x_my + 1;
          y_my = 1;
      }
}

void fix_boundary_tile_right(int itr, int x_start, int y_start)
{ 
      int x_my = x_start + 1;
      int y_my = y_start;
      //cout << " debug:: "<< x_my << "," << y_my << "\n";
      for (int j = 2; j <= tileLength; ++j)
      {   
          //#pragma omp parallel for
          for (int k = 1; k <= (j-1); ++k)
          {
            //cout << " debug:: "<< x_my << "," << y_my << "\n";
            ExpressionCPU;
            y_my = y_my - 1;
          }
          x_my = x_my + 1;
          y_my = y_start;
      }
}
