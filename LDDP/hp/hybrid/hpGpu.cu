#include "hip/hip_runtime.h"

#include "headers/myHeaders.h"

using namespace std;


int main(int argc, char const *argv[])
{	
	// Original array is 2d array : input image / matrix / DP table
	arrayOrg=new int *[nRows];
	for(int z=0 ; z<nRows ; z++)
	{
		arrayOrg[z]=new int[nCols];
	}

	// initialiation : not required if we are inputting the image : can be replaced by input code
	for (int i = 0; i < nRows; ++i)
	{	
		for (int j = 0; j < nCols; ++j)
		{
			arrayOrg[i][j] = rand() % 10 ;
		}
	}

	//flatten the array to 1d array
	 h_array = (int*)calloc(nRows*nCols, sizeof(int));
	  for (int i = 0; i < nRows; ++i)
	  {
	    for (int j = 0; j < nCols; ++j)
	    {
	     h_array[i*nCols + j] = arrayOrg [i][j];
	    }
	  }

	//copy the h_array to gpu
	int *d_array;
	hipMalloc((void**) &d_array, sizeof(int)*(nRows*nCols));
    hipMemcpy(d_array, h_array,sizeof(int)*(nRows*nCols), hipMemcpyHostToDevice);

    //configure kernel
	configure_kernal(nCols);
	
	//Execute on GPU 
	struct timeval start, end;
 	gettimeofday(&start, NULL);
	double time1 = omp_get_wtime();
	
	
   for (int i = 1; i < nRows; ++i)
   {
	   	update_array_gpu<<<dim3(g,1,1), dim3(x,1,1)>>>(i, nCols, d_array);
   }

	gettimeofday(&end, NULL);
  	double run_time = ((end.tv_sec - start.tv_sec)*1000 + (end.tv_usec - start.tv_usec)/1000.0);
	printf("Time (hp-gpu): %.3lf\n", run_time);
	cout <<"Time (hp-gpu): " <<(omp_get_wtime() - time1)*1000<< "\n";
	//cout << "Result on gpu: " << h_array [nRows*nCols - 1] << "\n";

	//copy back to cpu
	hipMemcpy(h_array, d_array,sizeof(int)*(nRows*nCols), hipMemcpyDeviceToHost);

	//Access the resultant matrix : dump into output file
	
	/*ofstream myfile ("../output_p.txt");
	for (int i = 0; i < nRows; ++i)
	{	
		for (int j = 0; j < nCols; ++j)
		{
			myfile << h_array[i*nCols + j] << "\t";
		}
		myfile << "\n";
	}*/
	
	return 0;
}



__global__ void update_array_gpu(int i, int numberOfThreadsRequired, int *d_array )
{
   long j=blockIdx.x *blockDim.x + threadIdx.x + 1;
   
   if (j>= numberOfThreadsRequired)
      {}
   else
   {
       GPU_Expression;   
   }
}

void configure_kernal(long numberOfThreadsRequired)
{
   if (numberOfThreadsRequired <= BLOCK_SIZE)
      {x=numberOfThreadsRequired ; g=1;}
   else
      {
         g= (numberOfThreadsRequired / BLOCK_SIZE ) + 1; x= BLOCK_SIZE;
      }

}