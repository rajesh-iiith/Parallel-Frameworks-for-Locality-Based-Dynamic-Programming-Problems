
#include "headers/myHeaders.h"

using namespace std;


int main(int argc, char const *argv[])
{	
	// Original array is 2d array : input image / matrix / DP table
	arrayOrg=new int *[nRows];
	for(int z=0 ; z<nRows ; z++)
	{
		arrayOrg[z]=new int[nCols];
	}

	// initialiation : not required if we are inputting the image : can be replaced by input code
	for (int i = 0; i < nRows; ++i)
	{	
		for (int j = 0; j < nCols; ++j)
		{
			arrayOrg[i][j] = rand() % 10 ;
		}
	}

	//flatten the array to 1d array
	 h_array = (int*)calloc(nRows*nCols, sizeof(int));
	  for (int i = 0; i < nRows; ++i)
	  {
	    for (int j = 0; j < nCols; ++j)
	    {
	     h_array[i*nCols + j] = arrayOrg [i][j];
	    }
	  }

	
	
	//Execute on CPU
	struct timeval start, end;
 	gettimeofday(&start, NULL);
	double time1 = omp_get_wtime();
	
	
   for (int i = 1; i < nRows; ++i)
   {
   		update_array_cpu(i);
   }

	gettimeofday(&end, NULL);
  	double run_time = ((end.tv_sec - start.tv_sec)*1000 + (end.tv_usec - start.tv_usec)/1000.0);
	printf("Time (hp-cpu): %.3lf\n", run_time);
	cout <<"Time (hp-cpu): " <<(omp_get_wtime() - time1)*1000<< "\n";
	//cout << "Result on cpu: " << h_array [nRows*nCols - 1] << "\n";

	//copy back to cpu
	//hipMemcpy(h_array, d_array,sizeof(int)*(nRows*nCols), hipMemcpyDeviceToHost);

	//Access the resultant matrix : dump into output file
	/*ofstream myfile ("../output_s.txt");
	for (int i = 0; i < nRows; ++i)
	{	
		for (int j = 0; j < nCols; ++j)
		{
			myfile << h_array[i*nCols + j] << "\t";
		}
		myfile << "\n";
	}*/
	
	return 0;
}


void update_array_cpu(int i)
{
		#pragma omp parallel for
		for (int j = 1; j < nRows ; ++j)
	   	{	
			CPU_Expression;
	   	}
}

